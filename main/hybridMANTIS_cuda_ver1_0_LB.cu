#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//
// 			     //////////////////////////////////////////////////////////
//  			     //							     //
// 			     //   	        hybridMANTIS v1.0		     //
// 			     //              fastDETECT2 - CUDA code                 //
//			     //		   (optical photons transport)		     //
//  			     //							     //
//                           //               used for Load Balancing                //
//			     //							     //
//			     //////////////////////////////////////////////////////////
//
// 
//
//
// ****Disclaimer****
//  This software and documentation (the "Software") were developed at the Food and Drug Administration (FDA) by employees of the Federal Government in
//  the course of their official duties. Pursuant to Title 17, Section 105 of the United States Code, this work is not subject to copyright protection
//  and is in the public domain. Permission is hereby granted, free of charge, to any person obtaining a copy of the Software, to deal in the Software
//  without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, or sell copies of the
//  Software or derivatives, and to permit persons to whom the Software is furnished to do so. FDA assumes no responsibility whatsoever for use by other
//  parties of the Software, its source code, documentation or compiled executables, and makes no guarantees, expressed or implied, about its quality,
//  reliability, or any other characteristic. Further, use of this code in no way implies endorsement by the FDA or confers any advantage in regulatory
//  decisions. Although this software can be redistributed and/or modified freely, we ask that any derivative works bear some notice that they are
//  derived from it, and any modified versions bear some notice that they have been modified. 
//
//	Detailed comments are available in "hybridMANTIS_cuda_ver1_0.cu" and "hybridMANTIS_c_ver1_0.c" files.
//
//	Associated publication: Sharma Diksha, Badal Andreu and Badano Aldo, "hybridMANTIS: a CPU-GPU Monte Carlo method for modeling indirect x-ray detectors with
//				columnar scintillators". Physics in Medicine and Biology, 57(8), pp. 2357–2372 (2012)
//
//
//	File:   	hybridMANTIS_cuda_ver1_0_LB.cu 			
//	Author: 	Diksha Sharma (US Food and Drug Administration)
//	Email: 		diksha.sharma@fda.hhs.gov			
//	Last updated:  	Apr 13, 2012
// 
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

/////////////////////////////////////////
//
//      Header libraries
//
/////////////////////////////////////////

#include <gsl/gsl_rng.h>
#include <gsl/gsl_randist.h>

#ifdef USING_CUDA
	#include <cutil_inline.h>
	#include <hip/hip_vector_types.h>
	#include <stdint.h>
#endif

/////////////////////////////////////////
//
//      Global variables
//
/////////////////////////////////////////

#define max_photon_per_EDE 900000	// maximum number of optical photons that can be generated per energy deposition event (EDE)

#ifdef USING_CUDA
	#define gpubufsizeT 2304000	// GPU buffer size: # of events sent to the GPU
#endif

/////////////////////////////////////////
//
//      Include kernel program
//
/////////////////////////////////////////
#include "kernel_cuda_c_ver1_0_LB.cu"

/////////////////////////////////////////
//
//      CUDA parameters
//
/////////////////////////////////////////
#ifdef USING_CUDA
	#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
	printf("Error at %s:%d\n",__FILE__,__LINE__); \
	return EXIT_FAILURE;}} while(0)

	#define GRIDSIZE 18000		// number of blocks
	#define BLOCKSIZE 128		// number of threads
#endif




////////////////////////////////////////////////////////////////////////////
//				MAIN PROGRAM			          //
////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// gpuoptlb():   Performs optical transport for finding optimal load using load balancing in the GPU 
//	  	 Input arguments: gptime, lbbuf
//
//		 gptime: time taken by GPU to call this routine
//		 lbbuf:  GPU buffer size defined by user in PENELOPE tally code. Only to be used for load balancing. This is different from 'gpubufsize'.
//
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	extern "C" void gpuoptlb_(double *gptime, int *lbbuf)
	{    

		float xdetector, ydetector, radius, height, n_C, n_IC, top_absfrac, bulk_abscoeff, beta, d_min, lbound_x, lbound_y, ubound_x, ubound_y, d_max, yield, sensorRefl;
		int pixelsize, num_primary, min_optphotons, max_optphotons, num_bins;
		dim3 threads, blocks;
                float gpuTime = 0.0f;
		int devID;
		double mu_gsl;	
		unsigned long long int host_num_generated = 0, host_num_detect = 0;
		unsigned long long int host_num_lost = 0;
		unsigned long long int host_num_abs_top = 0, host_num_abs_bulk = 0;
		unsigned long long int host_num_outofcol = 0;
		unsigned long long int host_num_theta1 = 0;
		const gsl_rng_type * Tgsl;
		gsl_rng * rgsl;
		int xdim = 0;
		int ydim = 0;
		int indexi=0, indexj=0;
		int my_index=0;
		size_t pitch;
		int nbytes = (*lbbuf)*sizeof(struct start_info);

		// allocate memory pointers
		unsigned long long int *myimage = 0;	// device memory for output image
		unsigned long long int *h_myimage = 0; 	// host memory for output image     
		int *num_detected_primary = 0;		// device memory for # detected photons/primary
		int *h_num_detected_primary = 0;		// host memory to get # detected/primary
		struct start_info *h_a = 0;             // pointer to the struct info data in the host memory
		struct start_info *d_a = 0;             // pointers to struct data in the device memory


		// set the device with max GFlops	
		devID = cutGetMaxGflopsDeviceId();
		hipSetDevice( devID );

 		// copy to local variables from PENELOPE buffers
		xdetector = inputargs_.detx;		// x dimension of detector (in um). x in (0,xdetector)
		ydetector = inputargs_.dety;		// y dimension of detector (in um). y in (0,ydetector)
		height = inputargs_.detheight;		// height of column and thickness of detector (in um). z in range (-H/2, H/2)
		radius = inputargs_.detradius;		// radius of column (in um).
		n_C = inputargs_.detnC;			// refractive index of columns
		n_IC = inputargs_.detnIC;		// refractive index of intercolumnar material
		top_absfrac = inputargs_.dettop;	// column's top surface absorption fraction (0.0, 0.5, 0.98)
		bulk_abscoeff = inputargs_.detbulk;	// column's bulk absorption coefficient (in um^-1) (0.001, 0.1 cm^-1) 
		beta = inputargs_.detbeta;		// roughness coefficient of column walls
		d_min = inputargs_.detdmin;		// minimum distance a photon can travel when transmitted from a column
		d_max = inputargs_.detdmax;
		lbound_x = inputargs_.detlboundx;	// x lower bound of region of interest of output image (in um)
		lbound_y = inputargs_.detlboundy;	// y lower bound (in um)
		ubound_x = inputargs_.detuboundx;	// x upper bound (in um) 
		ubound_y = inputargs_.detuboundy;	// y upper bound (in um)
		yield = inputargs_.detyield;		// yield (/eV)
		pixelsize = inputargs_.detpixel;	// 1 pixel = pixelsize microns (in um)
		sensorRefl = inputargs_.detsensorRefl;	// Non-Ideal sensor reflectivity (%)
		num_primary = inputargs_.mynumhist;	// total number of primaries to be simulated
		min_optphotons = inputargs_.minphotons;	// minimum number of optical photons detected to be included in PHS
		max_optphotons = inputargs_.maxphotons;	// maximum number of optical photons detected to be included in PHS
		num_bins = inputargs_.mynumbins;	// number of bins for genrating PHS
		
	      	// create a generator chosen by the 
		//  environment variable GSL_RNG_TYPE 
	       	gsl_rng_env_setup();	     
	       	Tgsl = gsl_rng_default;
	       	rgsl = gsl_rng_alloc (Tgsl);
	       	
	       	// dimensions of PRF image
		xdim = ceil((ubound_x - lbound_x)/pixelsize);
		ydim = ceil((ubound_y - lbound_y)/pixelsize);

		// allocate device memory for storing output arrays
		hipMallocPitch((void**)&myimage, &pitch, xdim*sizeof(unsigned long long int), ydim);		// allocate 2D image array
		cutilSafeCall( hipMemset2D(myimage, pitch, 0, xdim*sizeof(unsigned long long int), ydim) );	// initialize to 0
		cutilSafeCall( hipMalloc((void**)&num_detected_primary, sizeof(int)*num_primary) );		// outputting # detected/primary
		cutilSafeCall( hipMemset(num_detected_primary, 0, sizeof(int)*num_primary) );			// initialize to 0

		// allocate host and device memory for stroing interaction event buffer information
		cutilSafeCall( hipHostMalloc((void**)&h_a, nbytes) ); 
		cutilSafeCall( hipMalloc((void**)&d_a, nbytes) );

		// reset the host counters
		host_num_generated=0;
		host_num_detect=0;
		host_num_abs_top=0;	
		host_num_abs_bulk=0;	
		host_num_lost=0;
		host_num_outofcol=0;
		host_num_theta1=0;

		// reset device counters to zero
		cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_detectT"),&host_num_detect,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));	
		cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_generatedT"),&host_num_generated,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));	
		cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_abs_topT"),&host_num_abs_top,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));	
		cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_abs_bulkT"),&host_num_abs_bulk,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));	
		cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_lostT"),&host_num_lost,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));
		cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_outofcolT"),&host_num_outofcol,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));
		cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_theta1T"),&host_num_theta1,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));


		// synchronize threads to ensure that previous kernel has finished
		hipDeviceSynchronize();
    
		cutilSafeCall( hipHostMalloc((void**)&h_myimage, xdim*ydim*sizeof(unsigned long long int)) ); 
		cutilSafeCall( hipHostMalloc((void**)&h_num_detected_primary, sizeof(int)*num_primary) );

		for(indexj=0; indexj < num_primary; indexj++)
		  h_num_detected_primary[indexj] = 0;
		  
		int *h_histogram = 0;		// host memory for storing histogram of # photons detected/primary
		h_histogram = (int*)malloc(sizeof(int)*num_bins);
			
		for(indexj=0; indexj < num_bins; indexj++)
		  h_histogram[indexj] = 0;


		// assign number of threads and blocks
		threads = dim3(BLOCKSIZE,1);
		blocks = dim3(GRIDSIZE,1);

		// reading data from lbbuf
		for(my_index = 0; my_index < (*lbbuf); my_index++)		// iterate over x-rays
		{

			// units in the penelope output file are in cm. Convert to microns.
			h_a[my_index].str_x = optical_.xbufopt[my_index] * 10000.0f;	// x-coordinate of interaction event.
			h_a[my_index].str_y = optical_.ybufopt[my_index] * 10000.0f;	// y-coordinate
			h_a[my_index].str_z = optical_.zbufopt[my_index] * 10000.0f;	// z-coordinate
			h_a[my_index].str_E = optical_.debufopt[my_index];		// energy deposited
			h_a[my_index].str_histnum = optical_.nbufopt[my_index];		// x-ray history number

			// sample # optical photons based on light yield and energy deposited for this interaction event (using Poisson distribution)
			mu_gsl = (double)h_a[my_index].str_E * yield;
			h_a[my_index].str_N = gsl_ran_poisson(rgsl,mu_gsl);

			if(h_a[my_index].str_N > max_photon_per_EDE)
			{
				printf("\n\n GPU str_n exceeds max photons. program is exiting - %d !! \n\n", h_a[my_index].str_N);
				exit(0);
			}

		} // for loop ends

		    // create cuda event handles
		    hipEvent_t start, stop;
		    cutilSafeCall( hipEventCreate(&start) );
		    cutilSafeCall( hipEventCreate(&stop)  );
    

		    // execute the kernel 
			cutilSafeCall( cutilDeviceSynchronize() );
		
			hipEventRecord(start, 0);

			// asynchronously copy data from host to device	(all to stream 0)
			cutilSafeCall( hipMemcpyAsync(d_a, h_a, nbytes, hipMemcpyHostToDevice, 0) );

			// each kernel has BLOCKSIZE threads; each thread transports one event in the buffer (info.str_N optical photons)
			algoT<<<blocks, threads, 0, 0>>>(d_a, myimage, num_detected_primary, pitch, (*lbbuf), xdetector, ydetector, radius, height, n_C, n_IC, top_absfrac, bulk_abscoeff, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, d_max, sensorRefl); 
				
			// asynchronously copy image data from device to host
			cutilSafeCall( hipMemcpy2DAsync((void*)h_myimage,sizeof(unsigned long long int)*xdim,(void*)myimage,pitch,sizeof(unsigned long long int) *xdim,ydim,hipMemcpyDeviceToHost, 0) );
			cutilSafeCall( hipMemcpyAsync(h_num_detected_primary, num_detected_primary, sizeof(int)*num_primary, hipMemcpyDeviceToHost, 0) );

			hipEventRecord(stop, 0);

			 // have CPU do some work while waiting for stage 1 to finish
			 unsigned long int counter123=0;
			 while( hipEventQuery(stop) == hipErrorNotReady )
			 {
				counter123++;
			 }
			 cutilSafeCall( hipEventElapsedTime(&gpuTime, start, stop) );
			 *gptime = (double)(gpuTime*0.001);	// convert in sec
                                 
		         cutilCheckMsg("algo() execution failed\n");

		         hipDeviceSynchronize();	// to ensure that gpu finished before copying back the final results.


		// copy counters from device to host
		cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_detect,num_detectT,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));	
		cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_generated,num_generatedT,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));	
		cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_abs_top,num_abs_topT,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));	
		cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_abs_bulk,num_abs_bulkT,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));
		cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_lost,num_lostT,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));
		cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_outofcol,num_outofcolT,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));
		cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_theta1,num_theta1T,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));


		// add h_myimage to the new_myimage
		for(indexi = 0; indexi < ydim; indexi++)
		 for(indexj = 0; indexj < xdim; indexj++)
			outputimage_.newimageopt[indexi][indexj] = outputimage_.newimageopt[indexi][indexj] + h_myimage[indexi*xdim + indexj];

		// make histogram of number of detected photons/primary for num_bins
		int binsize=0, newbin=0;
		int bincorr=0;
							
		binsize = floor((max_optphotons-min_optphotons)/num_bins);	// calculate size of each bin. Assuming equally spaced bins.
		bincorr = floor(min_optphotons/binsize);			// correction in bin number if min_optphotons > 0.
			
		for(indexi = 0; indexi < num_primary; indexi++)
		 {
		 	newbin = floor(h_num_detected_primary[indexi]/binsize) - bincorr;	// find bin #
		 	
		 	if(h_num_detected_primary[indexi] > 0)	// store only non-zero bins
		 	{
		 		if(h_num_detected_primary[indexi] <= min_optphotons)	// # detected < minimum photons given by user, add to the 1st bin
			 		h_histogram[0]++;
			 	else if(h_num_detected_primary[indexi] >= max_optphotons)	// # detected > maximum photons given by user, then add to the last bin
			 		h_histogram[num_bins-1]++;
			 	else
				 	h_histogram[newbin]++; 
			}
		 }
			
		// add num_detected_primary to gldetprimary array in PENELOPE
		for(indexi = 0; indexi < num_bins; indexi++)
			outputdetprim_.gldetprimary[indexi] = outputdetprim_.gldetprimary[indexi] + h_histogram[indexi];
				
			   
		// type cast unsigned long long int to double
		double cast_host_num_generated;
		double cast_host_num_detect;
		double cast_host_num_abs_top;
		double cast_host_num_abs_bulk;
		double cast_host_num_lost;
		double cast_host_num_outofcol;
		double cast_host_num_theta1;
		double cast_gputime;

		cast_host_num_generated = (double)host_num_generated;
		cast_host_num_detect    = (double)host_num_detect;
		cast_host_num_abs_top   = (double)host_num_abs_top;
		cast_host_num_abs_bulk  = (double)host_num_abs_bulk;
		cast_host_num_lost      = (double)host_num_lost;
		cast_host_num_outofcol  = (double)host_num_outofcol;
		cast_host_num_theta1    = (double)host_num_theta1;
		cast_gputime		= (double)(gpuTime);

		 // save to global counters
		 optstats_.glgen      = optstats_.glgen      + cast_host_num_generated;
		 optstats_.gldetect   = optstats_.gldetect   + cast_host_num_detect;
		 optstats_.glabstop   = optstats_.glabstop   + cast_host_num_abs_top;
		 optstats_.glabsbulk  = optstats_.glabsbulk  + cast_host_num_abs_bulk;
		 optstats_.gllost     = optstats_.gllost     + cast_host_num_lost;
		 optstats_.gloutofcol = optstats_.gloutofcol + cast_host_num_outofcol;
		 optstats_.gltheta1   = optstats_.gltheta1   + cast_host_num_theta1;
		 optstats_.glgputime  = optstats_.glgputime  + cast_gputime;

	 
		 // release resources
		 cutilSafeCall(hipFree(d_a));
		 cutilSafeCall(hipFree(myimage));
		 cutilSafeCall(hipFree(num_detected_primary));
		 hipHostFree(h_a);
		 hipHostFree(h_myimage);
		 hipHostFree(h_num_detected_primary);
		 
    	         free(h_histogram);
		
		return;
	}	// CUDA main() ends
	
#endif
