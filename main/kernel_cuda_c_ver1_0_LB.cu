#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// 			     //////////////////////////////////////////////////////////
//  			     //							     //
// 			     //   	        hybridMANTIS v1.0		     //
// 			     //   	  fastDETECT2 kernel - CUDA + C  	     //
//			     //		   (optical photons transport)		     //
//  			     //							     //
//                           //               used for Load Balancing                //
//			     //							     //
//			     //////////////////////////////////////////////////////////
//
// 
//
// ****Disclaimer****
//  This software and documentation (the "Software") were developed at the Food and Drug Administration (FDA) by employees of the Federal Government in
//  the course of their official duties. Pursuant to Title 17, Section 105 of the United States Code, this work is not subject to copyright protection
//  and is in the public domain. Permission is hereby granted, free of charge, to any person obtaining a copy of the Software, to deal in the Software
//  without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, or sell copies of the
//  Software or derivatives, and to permit persons to whom the Software is furnished to do so. FDA assumes no responsibility whatsoever for use by other
//  parties of the Software, its source code, documentation or compiled executables, and makes no guarantees, expressed or implied, about its quality,
//  reliability, or any other characteristic. Further, use of this code in no way implies endorsement by the FDA or confers any advantage in regulatory
//  decisions. Although this software can be redistributed and/or modified freely, we ask that any derivative works bear some notice that they are
//  derived from it, and any modified versions bear some notice that they have been modified. 
//
//	Detailed comments included in "kernel_cuda_c_ver1_0.cu" file.
//
//	Associated publication: Sharma Diksha, Badal Andreu and Badano Aldo, "hybridMANTIS: a CPU-GPU Monte Carlo method for modeling indirect x-ray detectors with
//				columnar scintillators". Physics in Medicine and Biology, 57(8), pp. 2357–2372 (2012)
//
//
//
//	File:   	kernel_cuda_c_ver1_0_LB.cu 			
//	Author: 	Diksha Sharma (US Food and Drug Administration)
//	Email: 		diksha.sharma@fda.hhs.gov			
//	Last updated:  	Apr 13, 2012
// 
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


/////////////////////////////////////////
//
//      Header libraries
//
/////////////////////////////////////////

	#include <math.h>
	#include <stdio.h>
	#include <stdlib.h>
	#include <string.h>
	#include <sys/time.h>
	#include <time.h>

/////////////////////////////////////////
//
//       Constants
//
/////////////////////////////////////////

	#define twopipen 6.283185308	// 2*PI
	#define pi 3.14159265		// PI
	#define epsilon 8.1929093e-6	// a very small number for float comparisons


/////////////////////////////////////////////////////////////////////////////////////
//
//     Data structure for storing a scintillation event location and deposited energy
//
/////////////////////////////////////////////////////////////////////////////////////

	struct start_info
	{
		double str_x;		
		double str_y;		
		double str_z;		
		double str_E;		
		int str_histnum;	
		int str_N;		
	};



//////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//	Fortran structure declarations - using PENELOPE 2006 (coded in Fortran)
//	A 'common' block in Fortran needs to be declared here to allow calling function interexchangebly.	
// 	
//////////////////////////////////////////////////////////////////////////////////////////////////////////

// Similar structure to 'start_info' - declared in PENELOPE
	#ifdef USING_CUDA
		extern "C" struct
		{
			double xbufopt[gpubufsizeT];	
			double ybufopt[gpubufsizeT];	
			double zbufopt[gpubufsizeT];	
			double debufopt[gpubufsizeT];	
			int nbufopt[gpubufsizeT];	
			int myctropt;			
		        int cpu_num_real;		
		} optical_;
	#else
		extern struct
		{
			double xbufopt[mybufsizeT];
			double ybufopt[mybufsizeT];
			double zbufopt[mybufsizeT];
			double debufopt[mybufsizeT];	
			int nbufopt[mybufsizeT];		
			int myctropt;			
		        int cpu_num_real;			
		} optical_;
	#endif

// Storing optical output statistics - declared in PENELOPE
	#ifdef USING_CUDA
		extern "C" struct
		{
			double glgen;			
			double gldetect;		
			double glabstop;		
			double glabsbulk;		
			double gllost;			
			double gloutofcol;		
			double gltheta1;		
			double glgputime;		
		} optstats_;
	#else
		extern struct
		{
			double glgen;
			double gldetect;
			double glabstop;
			double glabsbulk;
			double gllost;
			double gloutofcol;
			double gltheta1;
			double glgputime;		
		} optstats_;
	#endif

// Storing deposited energy and # optical photons detected - declared in PENELOPE
	#ifdef USING_CUDA
		extern "C" struct
		{
			int gldetprimary[1000];		
		} outputdetprim_;
	#else
		extern struct
		{
			int gldetprimary[1000];
		} outputdetprim_;
	#endif

// Structure for storing point response functions - declared in PENELOPE
	#ifdef USING_CUDA
		extern "C" struct
		{
			unsigned long long int newimageopt[501][501];	
			unsigned long long int tempimageopt[501][501];
		} outputimage_;
	#else
		extern struct
		{
			unsigned long long int newimageopt[501][501];
			unsigned long long int tempimageopt[501][501];
		} outputimage_;
	#endif

// Storing the memory addresses of arrays, in order to call fastDETECT2 in the GPU asynchronously - declared in PENELOPE
	#ifdef USING_CUDA
		extern "C" struct
		{
			unsigned long long int gpuimage;		
			unsigned long long int gpudetect;		
			unsigned long long int hosta;			
			unsigned long long int deva;			
			unsigned long long int devpitch;			
		} gpumemaddr_;
	#endif

// Storing the input arguments - declared in PENELOPE
	#ifdef USING_CUDA
		extern "C" struct
		{
			double detx;		
			double dety;		
			double detheight;	
			double detradius;	
			double detnC;		
			double detnIC;		
			double dettop;		
			double detbulk;		
			double detbeta;		
			double detdmin;		
			double detdmax;		
			double detlboundx;	
			double detlboundy;	
			double detuboundx;	
			double detuboundy;	
			double detyield;	
			double detsensorRefl;	
			int detpixel;		
			int rungpu;
			int machinenum;
			int mynumhist;	
			int minphotons;	
			int maxphotons;	
			int mynumbins;	
		} inputargs_;
	#else
		extern struct
		{
			double detx;
			double dety;
			double detheight;
			double detradius;
			double detnC;
			double detnIC;
			double dettop;
			double detbulk;
			double detbeta;
			double detdmin;
			double detdmax;
			double detlboundx;
			double detlboundy;
			double detuboundx;
			double detuboundy;
			double detyield;
			double detsensorRefl;
			int detpixel;
			int rungpu;
			int machinenum;
			int mynumhist;	
			int minphotons;	
			int maxphotons;	
			int mynumbins;	
		} inputargs_;
	#endif

/////////////////////////////////////////
//
//       Function declarations
//
/////////////////////////////////////////

// transports optical photon from its generation until it ends (detected/absorbed/lost).
	#ifdef USING_CUDA
		__global__ void algoT(struct start_info *info, unsigned long long int *myimage, int *num_detected_primary, size_t pitch, int rowsread, float xdetector, 
		float ydetector, float R, float H, float n1, float n2, float top_absfrac, float bulk_abscoeff, float beta, float d_min, int pixelsize, float lbound_x, float lbound_y,
		float ubound_x, float ubound_y, float d_max, float sensorRefl);	
	#else
		int algoT(float *normal, float *old_pos, float *pos, float *dcos, unsigned long long int *num_rebound, int* seed, struct start_info info, 
		unsigned long long int *myimage, float xdetector, float ydetector, float R, float H, float n1, float n2, float top_absfrac, float bulk_abscoeff, float beta, 
		float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, float sensorRefl, float d_max, int ydim, int *h_num_detected_prim);
	#endif

// photon within a column. calculate if it gets absorbed or moves inside the column.
	#ifdef USING_CUDA
		__device__ inline int isotropicT(float3 *pos, float3 *dcos, int2* seed, float bulk_abscoeff, float R, float H, float xdetector, float ydetector,
	 	struct start_info *info, unsigned long long int mynum_rebound, float *XcT, float *YcT, int mytid);
	#else
		int isotropicT(float *pos, float *dcos, int* seed, float bulk_abscoeff, float R, float H, float xdetector, float ydetector, 
		struct start_info info, unsigned long long int mynum_rebound);
	#endif

// photon within a column. calculate distance to next position in the same column and move it.
	#ifdef USING_CUDA
		__device__ float dist_to_surfaceT(float3 *pos, float3 *dcos, float R, float H, float xdetector, float ydetector, struct start_info *info, 
		unsigned long long int mynum_rebound, float *XcT, float *YcT, int mytid);	
	#else
		float dist_to_surfaceT(float *pos, float *dcos, float R, float H, float xdetector, float ydetector, struct start_info info, 
		unsigned long long int mynum_rebound);
	#endif

// photon within/between columns. calculate if it gets reflected or transmitted.
	#ifdef USING_CUDA
		__device__ int boundary_analysisT(float3 *normal, float3 *pos, float3 *dcos, int2* seed, float xdetector, float ydetector, float R, float H, float n1, float n2, 
		float top_absfrac, float beta, float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, float *XcT,
		float *YcT, size_t pitch, struct start_info *info, int mytid, int *num_detected_primary, float d_max, float sensorRefl);	
	#else
		int boundary_analysisT(float *normal, float *pos, float *dcos, int* seed, float xdetector, float ydetector, float R, float H, float n1, float n2, float top_absfrac, 			float beta, float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, struct start_info info, 
		float d_max, float sensorRefl, int ydim, int *h_num_detected_prim);	
	#endif

// transmit photon to another column. calculates the new position when it transmits, build new column and move photon here.
	#ifdef USING_CUDA
		__device__ int transmitT(float3 *pos, float3 *dcos, float3 *normal, int2* seed, float xdetector, float ydetector, float H, float top_absfrac, float beta, float d_min,
		int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, size_t pitch, struct start_info *info, int mytid, 
		int *num_detected_primary, float d_max, float sensorRefl, int flagCCT);	
	#else
		int transmitT(float *pos, float *dcos, float *normal, int* seed, float xdetector, float ydetector, float H, float top_absfrac, float beta, float d_min, int pixelsize, 			float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, struct start_info info, float d_max, float sensorRefl, int ydim, 
		int flagCCT, int *h_num_detected_prim);	
	#endif

// called when photon reflects from sensor plane (bottom surface) of the detector, outside of any column.
	#ifdef USING_CUDA
		__device__ int refl_bottomT(float3 *pos, float3 *dcos, float3 *normal, float xdetector, float ydetector, int2* seed, float beta, float d_min, float H, float d_max);	
	#else
		int refl_bottomT(float *pos, float *dcos, float *normal, float xdetector, float ydetector, int* seed, float beta, float d_min, float H, float d_max);	
	#endif

// calculate dot product of two vectors to give cosine of angle between them.
	#ifdef USING_CUDA
		__device__ inline float dot_productT(float3 *aa, float3 *b);	
	#else
		float dot_productT(float *aa, float *b);	
	#endif

// determine if photon got detected at sensor plane or is reflected back within the column
	#ifdef USING_CUDA
		__device__ inline int detectionT(float3 *pos, float H, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage,
		size_t pitch, struct start_info *info, int mytid, int *num_detected_primary, float sensorRefl, float d_min, int2* seed, float3 *dcos, float3 *normal, 
		float bulk_abscoeff, float R, float xdetector, float ydetector, unsigned long long int mynum_rebound, float *XcT, float *YcT);  
	#else
		int detectionT(float *pos, float H, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, 
		struct start_info info, float sensorRefl, float d_min, int* seed, float *dcos, float *normal, float bulk_abscoeff, float R, float xdetector, float ydetector, 
		unsigned long long int mynum_rebound, int ydim, int *h_num_detected_prim); 
	#endif

// calculate directional cosines of reflected/refracted vector.
	#ifdef USING_CUDA
		__device__ inline void trans_dir_cosT(float3 *dcos, float3 *normal, float refl_theta, float trans_theta, int flag_ref, int mytid, 
		struct start_info *info);  
	#else
		void trans_dir_cosT(float *dcos, float *normal, float refl_theta, float trans_theta, int flag_ref, struct start_info info);
	#endif

// calculate new rough normal vector depending on value of 'beta' (roughness coefficient).
	#ifdef USING_CUDA
		__device__ inline void RoughSurfaceT(float3 *normal, int2* seed, float beta);  
	#else
		void RoughSurfaceT(float *normal, int* seed, float beta); 
	#endif



// RANECU pseudo random number generator
	#ifdef USING_CUDA
		__device__ inline void init_PRNGT(int history_batch, int histories_per_thread, int seed_input, int2* seed);
		__device__ inline int abMODmT(int m, int a, int s);
		__device__ inline float ranecuT(int2* seed);
	#else
		void init_PRNGT(int history_batch, int histories_per_thread, int seed_input, int* seed);
		int abMODmT(int m, int a, int s);
		float ranecuT(int* seed);
	#endif


/////////////////////////////////////////
//
//       Global variables
//
/////////////////////////////////////////

	#ifdef USING_CUDA
		// counters
		__device__ unsigned long long int num_generatedT; 	
		__device__ unsigned long long int num_detectT;	 	
		__device__ unsigned long long int num_abs_topT;	 	
		__device__ unsigned long long int num_abs_bulkT;	
		__device__ unsigned long long int num_lostT;	 	
		__device__ unsigned long long int num_outofcolT;	
		__device__ unsigned long long int num_theta1T;	 
		__device__ float photon_distanceT;     		 
	#else
		// counters
		unsigned long long int num_generatedT=0;	
		unsigned long long int num_detectT=0;	
		unsigned long long int num_abs_topT=0;	
		unsigned long long int num_abs_bulkT=0;	
		unsigned long long int num_lostT=0;	
		unsigned long long int num_outofcolT=0;	 
		unsigned long long int local_counterT=0;	 
		unsigned long long int num_theta1T=0;	

		//flags
		int absorbedT=0;	
		int detectT=0;		
		int bulk_absT=0;	

		float XcT=0.0f;		
		float YcT=0.0f;
		float photon_distanceT=0.0f; 

		FILE *fp1;
	#endif

/////////////////////////////////////////
//
//    Functions definition
//
/////////////////////////////////////////

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// transports optical photon from its generation until it ends (detected/absorbed/lost).
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__global__ void algoT(struct start_info *info, unsigned long long int *myimage, int *num_detected_primary, size_t pitch, int rowsread, float xdetector, float ydetector, 
	float R, float H, float n1, float n2, float top_absfrac, float bulk_abscoeff, float beta, float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, 
	float ubound_y, float d_max, float sensorRefl)
	{

		unsigned long long int local_counter = 0; // total number of photons terminated (either detected at bottom, absorbed at top or in the bulk)
		unsigned long long int local_num_generated = 0;
		unsigned long long int num_rebound=0;
		float3 dcos, normal, pos; 
		float rr=0.0f, theta=0.0f;
		float XcT=0.0f;				// center x,y of the current cylinder
		float YcT=0.0f;
		int tid = threadIdx.x + blockIdx.x * blockDim.x;	// thread Id

		// flags
		int absorbedT=0;			// flag for photons absorbed at the top surface of the detector
		int detectT=0;				// flag for photons detected at bottom of the detector
		int bulk_absT=0;			// flag for photons absorbed in the material of a column


	if(tid < rowsread)	// number of threads = number of rows read from pen output file, should simulate photons, rest do nothing
	{
		int NUM_EACH_THREAD = info[tid].str_N;			// number of photons to be simulated by each thread

		// Initialize variables
		dcos.x = 0.0f; dcos.y = 0.0f; dcos.z = 0.0f;
		normal.x = 0.0f; normal.y = 0.0f; normal.z = 0.0f;

		pos.x = info[tid].str_x; pos.y = info[tid].str_y; pos.z = info[tid].str_z;	// starting location given by the host

		int seed_input = 271828182+tid; // ranecu seed input
		int2 seed;

		// Initialize the RANECU generator in a position far away from the previous history:
		init_PRNGT(tid, 50000, seed_input, &seed);     

		// Initalize the device memory - dcos
		dcos.z = (ranecuT(&seed) * 2.0f) - 1.0f;	// generate random number between -1.0 and 1.0

		rr = sqrt(1.0f - dcos.z*dcos.z);
		theta = ranecuT(&seed) * twopipen;	// generate random number between 0 and 2pi
	
		dcos.x = rr*cos(theta);
		dcos.y = rr*sin(theta);

	
		if (((sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z)) < (1.0f - epsilon)) || ((sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z)) > (1.0f + epsilon)))
		 {
			dcos.x = dcos.x/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
			dcos.y = dcos.y/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
			dcos.z = dcos.z/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
		 }

	   local_num_generated++;		// first particle generated for this thread

	   while(local_num_generated < (NUM_EACH_THREAD+1))	//run until NUM_EACH_THREAD particles generated
	     {

		if(absorbedT == 0)        
		 {
			bulk_absT = isotropicT(&pos, &dcos, &seed, bulk_abscoeff, R, H, xdetector, ydetector, &info[tid], num_rebound, &XcT, &YcT, tid);

			if(bulk_absT == 0)
			{
				detectT = detectionT(&pos, H, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, pitch, info, tid, num_detected_primary, sensorRefl, d_min, &seed, &dcos, &normal, bulk_abscoeff, R, xdetector, ydetector, num_rebound, &XcT, &YcT);
			}
		 }

	 
		if( ((detectT == 1) || (absorbedT == 1) || (bulk_absT == 1)) && (local_counter < (NUM_EACH_THREAD-1)) ) // particle terminated
		 {

			local_counter++;

			// re-initialize all the arrays
			dcos.z = (ranecuT(&seed) * 2.0f) - 1.0f;	// generate random number between -1.0 and 1.0

			rr = sqrt(1.0f - dcos.z*dcos.z);
			theta = ranecuT(&seed) * twopipen;	// generate random number between 0 and 2pi
	
			dcos.x = rr*cos(theta);
			dcos.y = rr*sin(theta);

			if (((sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z)) < (1.0f - epsilon)) || ((sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z)) > (1.0f + epsilon)))
			 {
				dcos.x = dcos.x/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
				dcos.y = dcos.y/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
				dcos.z = dcos.z/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
			 }


			// set starting location of photon
			pos.x = info[tid].str_x; pos.y = info[tid].str_y; pos.z = info[tid].str_z;
			normal.x = 0.0f; normal.y = 0.0f; normal.z = 0.0f;
	
			if(beta > 0.0f)
				RoughSurfaceT(&normal, &seed, beta);	// new normal for rough surface

			local_num_generated++;
			absorbedT = 0;
			detectT = 0;
			bulk_absT = 0;
			num_rebound = 0;
			XcT = 0.0f;
			YcT = 0.0f;

		 }
		else if( ((detectT == 1) || (absorbedT == 1) || (bulk_absT == 1)) && (local_counter == (NUM_EACH_THREAD-1)) )
		 {
			local_counter++;
			break;
	
		 }
		else if( (detectT == 0) && (absorbedT == 0) && (bulk_absT == 0) && (fabs(dcos.z - 0.0f) < epsilon) )  // checking for trapped particle going back and forth with dcos(z)=0
		 {
			// kill the particle and generate a new one instead - do not increment the counter

			// re-initialize all the arrays
	 		dcos.z = (ranecuT(&seed) * 2.0f) - 1.0f;	// generate random number between -1.0 and 1.0

			rr = sqrt(1.0f - dcos.z*dcos.z);
			theta = ranecuT(&seed) * twopipen;	// generate random number between 0 and 2pi
	
			dcos.x = rr*cos(theta);
			dcos.y = rr*sin(theta);

			if (((sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z)) < (1.0f - epsilon)) || ((sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z)) > (1.0f + epsilon)))
			 {
				dcos.x = dcos.x/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
				dcos.y = dcos.y/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
				dcos.z = dcos.z/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
			 }

			// set starting location of photon
			pos.x = info[tid].str_x; pos.y = info[tid].str_y; pos.z = info[tid].str_z;

			normal.x = 0.0f; normal.y = 0.0f; normal.z = 0.0f;
	
			if(beta > 0.0f)
				RoughSurfaceT(&normal, &seed, beta);	// new normal for rough surface

			absorbedT = 0;
			detectT = 0;
			bulk_absT = 0;
			num_rebound = 0;
			XcT = 0.0f;
			YcT = 0.0f;
		 }
		else
		 {
			num_rebound++;
		    	absorbedT = boundary_analysisT(&normal, &pos, &dcos, &seed, xdetector, ydetector, R, H, n1, n2, top_absfrac, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, &XcT, &YcT, pitch, info, tid, num_detected_primary, d_max, sensorRefl);

		 }

	  } // while loop ends

	atomicAdd(&num_generatedT, local_num_generated);

	}	// if tid=rowsread ends

	 return;
	}
#else
	int algoT(float *normal, float *old_pos, float *pos, float *dcos, unsigned long long int *num_rebound, int* seed, struct start_info info,
	unsigned long long int *myimage, float xdetector, float ydetector, float R, float H, float n1, float n2, float top_absfrac, float bulk_abscoeff, float beta, float d_min, 
	int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, float sensorRefl, float d_max, int ydim, int *h_num_detected_prim)
	{


		float rr=0.0f, theta=0.0f;
		float norm = 0.0f;
		float rnd_num = 0.0f;
		int myresult = 0;


		if(absorbedT == 0)        
		 {
			bulk_absT = isotropicT(pos, dcos, seed, bulk_abscoeff, R, H, xdetector, ydetector, info, num_rebound[local_counterT]);

			if(bulk_absT == 0)
			{
				detectT = detectionT(pos, H, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, info, sensorRefl, d_min, seed, dcos, normal, bulk_abscoeff, R, xdetector, ydetector, num_rebound[local_counterT], ydim, h_num_detected_prim);
			}
		 }

	 
		if( (detectT == 1) || (absorbedT == 1) || (bulk_absT == 1) )
		 {
			local_counterT++;

			// re-initialize all the arrays
			rnd_num = (ranecuT(seed) * 2.0f) - 1.0f; 
	 	
			while(fabs(rnd_num) <= 0.01)	
			 {
		   		rnd_num = (ranecuT(seed) * 2.0f) - 1.0f;  	
		 	 }

			dcos[2] = rnd_num;		// random number between (-1,1)
			rr = sqrt(1.0-rnd_num*rnd_num);
			theta=ranecuT(seed)*twopipen;
			dcos[0]=rr*cos(theta);
			dcos[1]=rr*sin(theta);

			norm = sqrt(dcos[0]*dcos[0] + dcos[1]*dcos[1] + dcos[2]*dcos[2]);

			if ((norm < (1.0f - epsilon)) || (norm > (1.0f + epsilon)))
			 {
				dcos[0] = dcos[0]/norm;
				dcos[1] = dcos[1]/norm;
				dcos[2] = dcos[2]/norm;
			 }


			// set starting location of photon
			pos[0] = info.str_x; pos[1] = info.str_y; pos[2] = info.str_z;
			old_pos[0] = info.str_x; old_pos[1] = info.str_y; old_pos[2] = info.str_z;

			normal[0] = 0.0f; normal[1] = 0.0f; normal[2] = 0.0f;
	
			if(beta > 0.0f)
			  RoughSurfaceT(normal, seed, beta);	// new normal for rough surface

			absorbedT = 0;
			detectT = 0;
			bulk_absT = 0;

			myresult = 1;
	
		 }
		else if( (detectT == 0) && (absorbedT == 0) && (bulk_absT == 0) && (fabs(dcos[2] - 0.0f) < epsilon) )  // checking for trapped particle going back and forth with dcos(z)=0
		 {
			// kill the particle and generate a new one instead - do not increment the counter

			// re-initialize all the arrays
			rnd_num = (ranecuT(seed) * 2.0f) - 1.0f; 
	 	
			while(fabs(rnd_num) <= 0.01)	
			 {
		   		rnd_num = (ranecuT(seed) * 2.0f) - 1.0f;  	
		 	 }

			dcos[2] = rnd_num;		// random number between (-1,1)
			rr = sqrt(1.0-rnd_num*rnd_num);
			theta=ranecuT(seed)*twopipen;
			dcos[0]=rr*cos(theta);
			dcos[1]=rr*sin(theta);

			norm = sqrt(dcos[0]*dcos[0] + dcos[1]*dcos[1] + dcos[2]*dcos[2]);

			if ((norm < (1.0f - epsilon)) || (norm > (1.0f + epsilon)))
			 {
				dcos[0] = dcos[0]/norm;
				dcos[1] = dcos[1]/norm;
				dcos[2] = dcos[2]/norm;
			 }

			// set starting location of photon
			pos[0] = info.str_x; pos[1] = info.str_y; pos[2] = info.str_z;

			normal[0] = 0.0f; normal[1] = 0.0f; normal[2] = 0.0f;
	
			if(beta > 0.0f)
				RoughSurfaceT(normal, seed, beta);	// new normal for rough surface

			absorbedT = 0;
			detectT = 0;
			bulk_absT = 0;

			myresult = 0;
		 }
		else
		 {
			num_rebound[local_counterT]++;
		    	absorbedT = boundary_analysisT(normal, pos, dcos, seed, xdetector, ydetector, R, H, n1, n2, top_absfrac, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, info, d_max, sensorRefl, ydim, h_num_detected_prim);

	 		myresult = 0;

		 }

	 return myresult;

	}
#endif


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// determine where photon hits next within the column or if it gets absorbed in the material
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ inline int isotropicT(float3 *pos, float3 *dcos, int2* seed, float bulk_abscoeff, float R, float H, float xdetector, float ydetector,
	struct start_info *info, unsigned long long int mynum_rebound, float *XcT, float *YcT, int mytid)
	{
		float dsurf = 999.0f;
		float dabs = 999.0f;
		int flag_bulkabs = 0;

		dsurf = dist_to_surfaceT(pos, dcos, R, H, xdetector, ydetector, info, mynum_rebound, XcT, YcT, mytid);	// distance to surface

		if (bulk_abscoeff > 0.0f)	
			dabs = (-1.0f/bulk_abscoeff) * log(ranecuT(seed));					// distance to absorption
		else
			dabs = 999999.0f;

		if (fabs(dsurf-(-99.0f)) < epsilon)				// particle lost because it went out of limit in dist_to_surface()
		{
			flag_bulkabs = 1;
		}
		else if ( (dsurf < dabs) && (dsurf >= 0.0f) )
		 {		
			flag_bulkabs = 0;
		 }
		else if ( (dsurf >= dabs) && (dabs >= 0.0f) )
		 {
			flag_bulkabs = 1;

			atomicAdd(&num_abs_bulkT,1);
		 }

	   return flag_bulkabs;
	}
#else
	int isotropicT(float *pos, float *dcos, int* seed, float bulk_abscoeff, float R, float H, float xdetector, float ydetector,struct start_info info,
	unsigned long long int mynum_rebound)
	{
		float dsurf = 999.0f;
		float dabs = 999.0f;
		int flag_bulkabs = 0;

		dsurf = dist_to_surfaceT(pos, dcos, R, H, xdetector, ydetector, info, mynum_rebound);	// distance to surface

		if (bulk_abscoeff > 0.0f)	
			dabs = (-1.0f/bulk_abscoeff) * log(ranecuT(seed));					// distance to absorption
		else
			dabs = 999999.0f;

		if (fabs(dsurf-(-99.0f)) < epsilon)				// particle lost because it went out of limit in dist_to_surface()
		{
			flag_bulkabs = 1;
		}
		else if ( (dsurf < dabs) && (dsurf >= 0.0f) )
		 {		
			flag_bulkabs = 0;
		 }
		else if ( (dsurf >= dabs) && (dsurf >= 0.0f) )
		 {
			flag_bulkabs = 1;

			num_abs_bulkT++;
		 }

	   return flag_bulkabs;
	}
#endif


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// calculate distance to surface (within same column)
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ float dist_to_surfaceT(float3 *pos, float3 *dcos, float R, float H, float xdetector, float ydetector, struct start_info *info, 
	unsigned long long int mynum_rebound, float *XcT, float *YcT, int mytid)
	{
		float d=999.0f;
		float d1=999.0f, d2=999.0f;
		float d_plane=999.0f, d_cyl=999.0f;
		float3 temp_pos = {0.0f};
		float3 my1 = {0.0f};
		float R1 = 999.0f, R2 = 999.0f;
		float stepsize = 0.5f;
		int repeat = 0, ctr1 = 0; 	// number of times photon should be moved in steps towards the column before killing. 
						// Valid only when goes out of column.

		temp_pos.x = pos->x;
		temp_pos.y = pos->y;
		temp_pos.z = pos->z;

		// center of first column (assumed as x,y position of the energy deposition from Penelope)
		if(mynum_rebound == 0)				
		{
			*XcT = info->str_x;
			*YcT = info->str_y;
		}

		// solving quadratic equation for distance from a point to the surface of cylinder
		my1.x = dcos->x*dcos->x + dcos->y*dcos->y;
		my1.y = 2.0f*( ((pos->x-(*XcT))*dcos->x) + ((pos->y-(*YcT))*dcos->y) );
		my1.z = (pos->x-(*XcT))*(pos->x-(*XcT)) + (pos->y-(*YcT))*(pos->y-(*YcT)) - (R*R);
	
		// actual distance d = (d1 or d2)/sin_theta2	
		d1 = (-my1.y + (sqrt( (my1.y*my1.y) - (4.0f*my1.x*my1.z) )))/(2.0f * my1.x);
		d2 = (-my1.y - (sqrt( (my1.y*my1.y) - (4.0f*my1.x*my1.z) )))/(2.0f * my1.x);


		// might hit either upper half surface or top
		if(dcos->z > 0.0f) 
		 {

		  if((fabs(dcos->x - 0.0f) < epsilon) && (fabs(dcos->y - 0.0f) < epsilon) && (fabs(dcos->z - 1.0f) < epsilon))  
		  // if particle travel straight in +z axis direction
		   {
			d = (H/2.0f - pos->z)/dcos->z;

			pos->z = H/2.0f;	

			pos->x = temp_pos.x + d*dcos->x;
			pos->y = temp_pos.y + d*dcos->y;
		   }
		  else
		   {		

			// calculate distance to infinite plane at z=H/2
			d_plane = (H/2.0f - pos->z)/(dcos->z);

			// calculate the distance to the upper half of the cylinder
			if(d1 >= d2) 
			{
				d_cyl = d1;
			}
			else if(d2 > d1)
			{
				d_cyl = d2;
			}

			// find min from d_plane and d_cyl
			if(d_plane >= d_cyl)
			 {
				d = d_cyl;
				pos->z = temp_pos.z + d*dcos->z;
			 }
			else
			 {
				d = d_plane;		
				pos->z = H/2.0f;
			 }

			pos->x = temp_pos.x + d*dcos->x;
			pos->y = temp_pos.y + d*dcos->y;

		   }	// else loop ends
	
		
		 } // if loop for dcos.z > 0 ends

		else if(dcos->z < 0.0f) // might hit either lower half or bottom of cylinder
		 {
		  // if particle travels in -Z direction staright, then it should get detected
		  if ((fabs(dcos->x-0.0f) < epsilon) && (fabs(dcos->y-0.0f) < epsilon) && (fabs(dcos->z - (-1.0f)) < epsilon))  
		   {
			d = (-H/2.0f - pos->z)/dcos->z;  
			pos->z = -H/2.0f;
				
			pos->x = temp_pos.x + d*dcos->x;
			pos->y = temp_pos.y + d*dcos->y;
		   }
		  else
		   {

			// calculate distance to infinite plane at z=-H/2
			d_plane = (-H/2.0f - pos->z)/(dcos->z);

			// calculate the distance to the lower half of the cylinder
			if(d1 >= d2) 
			{
				d_cyl = d1;
			}
			else if(d2 > d1)
			{
				d_cyl = d2;
			}
		

			// find min from d_plane and d_cyl
			if(d_plane >= d_cyl)
			 {
				d = d_cyl;
				pos->z = temp_pos.z + d*dcos->z;
			 }
			else
			 {
				d = d_plane;		
				pos->z = -H/2.0f;
			 }

			pos->x = temp_pos.x + d*dcos->x;
			pos->y = temp_pos.y + d*dcos->y;

		   }	// else loop ends

		 }	// else if loop for dcos.z < 0 ends

		else	// when dcos.z=0.0 (will hit only the side of the cylinder)
		 {
			// calculate the distance to the side of cylinder
			if(d1 >= d2) 
			{
				d_cyl = d1;
			}
			else if(d2 > d1)
			{
				d_cyl = d2;
			}
		
		
			d = d_cyl;
			pos->z = temp_pos.z + d*dcos->z;

			pos->x = temp_pos.x + d*dcos->x;
			pos->y = temp_pos.y + d*dcos->y;

		 }

		// condition to check that pos is within detector boundaries - if true, particle LOST
		if ( (pos->x < epsilon) || (pos->x > xdetector) || (pos->y < epsilon) || (pos->y > ydetector) || (pos->z < -H/2.0f) || (pos->z > H/2.0f)  )
			{
				d = -99.0f;
				atomicAdd(&num_lostT,1);
				goto distexit;
			}
		else
			atomicAdd(&photon_distanceT, d);		// add distance travelled to global variable


		// CHECK IF PHOTON IS OUTSIDE THE CURRENT COLUMN
		R1 = sqrt((pos->x - (*XcT))*(pos->x - (*XcT)) + (pos->y - (*YcT))*(pos->y - (*YcT)));

		// check if photon is out of current column
		repeat = 0;
		ctr1 = 0;

		while( (R1 > (R-1e-5)) && (repeat < 10) && (ctr1 < 10) ) // R1 > R1-some small value..because of single precision errors that comparison with R may generate
		{

			// store current position
			temp_pos.x = pos->x;
			temp_pos.y = pos->y;

			// move particle by 0.5 um in the incident direction
			pos->x = pos->x + stepsize*(-dcos->x);
			pos->y = pos->y + stepsize*(-dcos->y);

			R2 = sqrt((pos->x - (*XcT))*(pos->x - (*XcT)) + (pos->y - (*YcT))*(pos->y - (*YcT)));

			if(R2 > R1) // means the photon is moving farther away from the column 
			{	    // this can happen if the stepsize if too big and the photon passes through the column and gets out on other side.

				// move it back to previous position, reduce the stepsize and try moving it again.
				pos->x = temp_pos.x;
				pos->y = temp_pos.y;

				stepsize = stepsize/2.0f;
				ctr1++;
			}
			else
			{
				R1 = R2;
				repeat++;
			}

		}

		// kill the particle if still outside the column
		if(R1 > (R-1e-5))
		 {
			d = -99.0f;
			atomicAdd(&num_outofcolT,1);
			goto distexit;
		 }

		// condition to check that pos is within detector boundaries - if true, particle LOST
		if ( (pos->x < epsilon) || (pos->x > xdetector) || (pos->y < epsilon) || (pos->y > ydetector) || (pos->z < -H/2.0f) || (pos->z > H/2.0f)  )
			{
				d = -99.0f;
				atomicAdd(&num_lostT,1);
				goto distexit;
			}

	distexit:
	 return d;
	}	// CUDA dist_to_surface function ends
#else

	float dist_to_surfaceT(float *pos, float *dcos, float R, float H, float xdetector, float ydetector, struct start_info info, unsigned long long int mynum_rebound)
	{

		float d=999.0f;
		float d1=999.0f, d2=999.0f;
		float d_plane=999.0f, d_cyl=999.0f;
		float temp_pos[3] = {0.0f};
		float my1[3] = {0.0f};
		float R1 = 999.0f, R2 = 999.0f;
		float stepsize = 0.5f;
		int repeat = 0, ctr1 = 0; 	// number of times photon should be moved in steps towards the column before killing. 
					// Valid only when goes out of column.

		temp_pos[0] = pos[0];
		temp_pos[1] = pos[1];
		temp_pos[2] = pos[2];

		// center of first column (assumed as x,y position of the energy deposition from Penelope)
		if(mynum_rebound == 0)				
		{
			XcT = info.str_x;
			YcT = info.str_y;
		}

		// solving quadratic equation for distance from a point to the surface of cylinder
		my1[0] = dcos[0]*dcos[0] + dcos[1]*dcos[1];
		my1[1] = 2.0f*( ((pos[0]-(XcT))*dcos[0]) + ((pos[1]-(YcT))*dcos[1]) );
		my1[2] = (pos[0]-(XcT))*(pos[0]-(XcT)) + (pos[1]-(YcT))*(pos[1]-(YcT)) - (R*R);
	
		// actual distance d = (d1 or d2)/sin_theta2	
		d1 = (-my1[1] + (sqrt( (my1[1]*my1[1]) - (4.0f*my1[0]*my1[2]) )))/(2.0f * my1[0]);
		d2 = (-my1[1] - (sqrt( (my1[1]*my1[1]) - (4.0f*my1[0]*my1[2]) )))/(2.0f * my1[0]);

	
		// might hit either upper half surface or top
		if(dcos[2] > 0.0f) 
		 {

		  if((fabs(dcos[0] - 0.0f) < epsilon) && (fabs(dcos[1] - 0.0f) < epsilon) && (fabs(dcos[2] - 1.0f) < epsilon))  
		  // if particle travel straight in +z axis direction
		   {
			d = (H/2.0f - pos[2])/dcos[2];

			pos[2] = H/2.0f;	

			pos[0] = temp_pos[0] + d*dcos[0];
			pos[1] = temp_pos[1] + d*dcos[1];
		   }
		  else
		   {		

			// calculate distance to infinite plane at z=H/2
			d_plane = (H/2.0f - pos[2])/(dcos[2]);

			// calculate the distance to the upper half of the cylinder
			if(d1 >= d2) 
			{
				d_cyl = d1;
			}
			else if(d2 > d1)
			{
				d_cyl = d2;
			}

			// find min from d_plane and d_cyl
			if(d_plane >= d_cyl)
			 {
				d = d_cyl;
				pos[2] = temp_pos[2] + d*dcos[2];
			 }
			else
			 {
				d = d_plane;		
				pos[2] = H/2.0f;
			 }

			pos[0] = temp_pos[0] + d*dcos[0];
			pos[1] = temp_pos[1] + d*dcos[1];

		   }	// else loop ends
	
		
		 } // if loop for dcos[2] > 0 ends

		else if(dcos[2] < 0.0f) // might hit either lower half or bottom of cylinder
		 {
		  // if particle travels in -Z direction staright, then it should get detected
		  if ((fabs(dcos[0]-0.0f) < epsilon) && (fabs(dcos[1]-0.0f) < epsilon) && (fabs(dcos[2] - (-1.0f)) < epsilon))  
		   {
			d = (-H/2.0f - pos[2])/dcos[2];  
			pos[2] = -H/2.0f;
				
			pos[0] = temp_pos[0] + d*dcos[0];
			pos[1] = temp_pos[1] + d*dcos[1];
		   }
		  else
		   {

			// calculate distance to infinite plane at z=-H/2
			d_plane = (-H/2.0f - pos[2])/(dcos[2]);

			// calculate the distance to the lower half of the cylinder
			if(d1 >= d2) 
			{
				d_cyl = d1;
			}
			else if(d2 > d1)
			{
				d_cyl = d2;
			}
		

			// find min from d_plane and d_cyl
			if(d_plane >= d_cyl)
			 {
				d = d_cyl;
				pos[2] = temp_pos[2] + d*dcos[2];
			 }
			else
			 {
				d = d_plane;		
				pos[2] = -H/2.0f;
			 }

			pos[0] = temp_pos[0] + d*dcos[0];
			pos[1] = temp_pos[1] + d*dcos[1];

		   }	// else loop ends

		 }	// else if loop for dcos[2] < 0 ends

		else	// when dcos[2]=0.0 (will hit only the side of the cylinder)
		 {
			// calculate the distance to the side of cylinder
			if(d1 >= d2) 
			{
				d_cyl = d1;
			}
			else if(d2 > d1)
			{
				d_cyl = d2;
			}
		
		
			d = d_cyl;
			pos[2] = temp_pos[2] + d*dcos[2];

			pos[0] = temp_pos[0] + d*dcos[0];
			pos[1] = temp_pos[1] + d*dcos[1];

		 }

		// condition to check that pos is within detector boundaries - if true, particle LOST
		if ( (pos[0] < epsilon) || (pos[0] > xdetector) || (pos[1] < epsilon) || (pos[1] > ydetector) || (pos[2] < -H/2.0f) || (pos[2] > H/2.0f)  )
			{
				d = -99.0f;
				num_lostT++;
				goto distexit;
			}
		else
			photon_distanceT = photon_distanceT + d;		// add distance travelled to global variable


		// CHECK IF PHOTON IS OUTSIDE THE CURRENT COLUMN
		R1 = sqrt((pos[0] - XcT)*(pos[0] - XcT) + (pos[1] - YcT)*(pos[1] - YcT));

		// check if photon is out of current column
		repeat = 0;
		ctr1 = 0;

		while( (R1 > (R-1e-5)) && (repeat < 10) && (ctr1 < 10) ) // R1 > R1-some small value..because of single precision errors that comparison with R may generate
		{

			// store current position
			temp_pos[0] = pos[0];
			temp_pos[1] = pos[1];

			// move particle by 0.5 um in the incident direction
			pos[0] = pos[0] + stepsize*(-dcos[0]);
			pos[1] = pos[1] + stepsize*(-dcos[1]);

			R2 = sqrt((pos[0] - XcT)*(pos[0] - XcT) + (pos[1] - YcT)*(pos[1] - YcT));

			if(R2 > R1) // means the photon is moving farther away from the column 
			{	    // this can happen if the stepsize if too big and the photon passes through the column and gets out on other side.

				// move it back to previous position, reduce the stepsize and try moving it again.
				pos[0] = temp_pos[0];
				pos[1] = temp_pos[1];

				stepsize = stepsize/2.0f;
				ctr1++;
			}
			else
			{
				R1 = R2;
				repeat++;
			}


		}

		// kill the particle if still outside the column
		if(R1 > (R-1e-5))
		 {

			d = -99.0f;
			num_outofcolT++;
			goto distexit;
		 }

		// condition to check that pos is within detector boundaries - if true, particle LOST
		if ( (pos[0] < epsilon) || (pos[0] > xdetector) || (pos[1] < epsilon) || (pos[1] > ydetector) || (pos[2] < -H/2.0f) || (pos[2] > H/2.0f)  )
			{
				d = -99.0f;
				num_lostT++;
				goto distexit;
			}

	distexit:
	 return d;
	}	// C dist_to_surface function ends
#endif


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// calculate the directional cosines of the reflected vector
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ int boundary_analysisT(float3 *normal, float3 *pos, float3 *dcos, int2* seed, float xdetector, float ydetector, float R, float H, float n1, float n2, 
	float top_absfrac, float beta, float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, float *XcT, 
	float *YcT, size_t pitch, struct start_info *info, int mytid, int *num_detected_primary, float d_max, float sensorRefl)
	{
		float3 dcos_temp = {0.0f};
		float3 old_normal = {0.0f};
		float3 old_dcos = {0.0f};
		float Pr = 0.0f, Pt = 0.0f;	// Prob. of reflection and transmission
		float theta1 = 0.0f, theta2 = 0.0f;
		float temp_norm = 0.0f;
		float mag = 0.0f;
		float rr_rnd = 0.0f;
		float theta_rnd = 0.0f;
		float angle_oldN_R = 0.0f;
		float newdepthT = 0.0f;
		float cos_newangle = 0.0f;
		float newangle = 0.0f;
		float cct1 = 0.0f;	// columnar crosstalk
		int trans_flag = 0.0f;
		int flag_abs = 0;		// flag - particle got absorbed at top surface or exited during the transmission to another boundary
		int flag_call_transmit = 1;	// flag - particle is going to move within a column (flag = 0) [call isotropic()] or between columns (flag = 1) [call transmit()] 
		int reperturb_ctr = 0;
		int flagCCT = 0;	// flag to indicate in transmit() that the photon needs to cross over
		int theta1ctr=0;	// counter for theta1 > 90 degrees (max resampling 100 times)
		int oldN_Rctr=0;	// counter for angle_oldN_R > 90 degrees (max resampling 25 times)
		int newnormalctr=0;

		// determine the coordinates of normal
		if ( (fabs(pos->z - (float)(H/2.0f)) < epsilon) && (dcos->z > 0.0f) )	// reached top surface and dir. cosine in z-direction is positive
		{
	
			// top surface absorption - using absorption coefficient 'top_absfrac'
			if ( (top_absfrac > 0.0f) && (ranecuT(seed) < top_absfrac) )	// particle gets absorbed
			{
				flag_abs = 1;
				atomicAdd(&num_abs_topT, 1);
			}
			else
			{

				// specular reflector		
				normal->x = 0.0f;
				normal->y = 0.0f;
				normal->z = -1.0f;

				// assign new directional cosines
				dcos->z = -fabs((ranecuT(seed) * 2.0f) - 1.0f);
				rr_rnd = sqrt(1.0f - dcos->z*dcos->z);
				theta_rnd = ranecuT(seed)*twopipen;	

				dcos->x=rr_rnd*cos(theta_rnd);
				dcos->y=rr_rnd*sin(theta_rnd);
	
				flag_abs = 0;
			}

		}	
		else 	// compute the normal and check if gets reflected or transmitted
		{	
			// Columnar crosstalk
			newdepthT = H*0.2f;	// top 20% depth CCT=1. considering CsI layer only. NO organic polymer coating.
			
			if( (pos->z <= H/2.0f) && (pos->z >= (H/2.0f - newdepthT)) )	// top 20% - 100% cct
			{
				cct1 = 1.0f;
			}
			else if( (pos->z < (H/2.0f - newdepthT)) && (pos->z >= 0.0f) )  // from 20% depth to 50% - linear 100% to 50% 
			{
				cct1 = (pos->z/(2.0f*(H/2.0f - newdepthT))) + 0.5;	
			}
			else if( (pos->z < 0.0f) && (pos->z >= (-H/2.0f)) ) // bottom 50% to -H/2 - 50% to 100% CCT
			{
				cct1 = ( (pos->z - (-H/2.0f))/(2.0f * (-H/2.0f)) ) + 1.0 ;
			}
	
			if(ranecuT(seed) < cct1)		// columnar cross talk occurs
			{

				// photon crosses over to new column with random orientation. dcos do not change.
				flagCCT = 1;

				trans_flag = transmitT(pos, dcos, normal, seed, xdetector, ydetector, H, top_absfrac, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, pitch, info, mytid, num_detected_primary, d_max, sensorRefl, flagCCT);

				if (trans_flag == 1)		// photon exited
					flag_abs = 1;
				else if (trans_flag == 0)
				{
					// calculate new column's center coordinates
					*XcT = (float)( pos->x + R*(-normal->x) );
					*YcT = (float)( pos->y + R*(-normal->y) );

					flag_abs = 0;
				}
			}
			else
			{
			prpt:

				// within the column
				if(flag_call_transmit == 1)			// photon is currrently within a column with center Xc,Yc
				{
					mag = sqrt( (((*XcT)-pos->x) * ((*XcT)-pos->x)) + (((*YcT)-pos->y) * ((*YcT)-pos->y)) );
					normal->x = ((*XcT)-pos->x)/mag;
					normal->y = ((*YcT)-pos->y)/mag;
					normal->z = 0.0f;
		
					if(beta > 0.0f)
						RoughSurfaceT(normal, seed, beta);	// new normal for rough surface

					flag_abs = 0;
				}
				// outside the column
				else if (flag_call_transmit == 0)		// photon is currently between columns and has not entered any column yet. New normal is sampled in the transmit(), so do not calculate normal here.
				{
					// center of new column (obtained by inverting the new normal sampled in transmit() and finding center at distance R from current position)
					*XcT = (float)( pos->x + R*(-normal->x) );
					*YcT = (float)( pos->y + R*(-normal->y) );

				        flag_abs = 0;
				}

				// -dcos -> inverted the incident vector to get the smaller angle, else would have to do angle = 180-angle
				dcos_temp.x = -dcos->x;
				dcos_temp.y = -dcos->y;
				dcos_temp.z = -dcos->z;

				old_normal.x = normal->x;
				old_normal.y = normal->y;
				old_normal.z = normal->z;
	
				old_dcos.x = dcos->x;
				old_dcos.y = dcos->y;
				old_dcos.z = dcos->z;

			reperturb:
				normal->x = old_normal.x;
				normal->y = old_normal.y;
				normal->z = old_normal.z;

				dcos->x = old_dcos.x;
				dcos->y = old_dcos.y;
				dcos->z = old_dcos.z;

				dcos_temp.x = -dcos->x;
				dcos_temp.y = -dcos->y;
				dcos_temp.z = -dcos->z;

				if( (flag_call_transmit == 1) && (reperturb_ctr != 0) )
				 {
					if(beta > 0.0f)
						RoughSurfaceT(normal, seed, beta);	// new normal for rough surface
				 }
				if( (flag_call_transmit == 0) && (reperturb_ctr != 0) )
				 {
					if(beta > 0.0f)
						RoughSurfaceT(normal, seed, beta);	// the sampled normal for transmitted photon needs to be perturbed

					// center of new column (obtained by inverting the new normal sampled in transmit() and finding center at distance R from current position)
					*XcT = (float)( pos->x + R*(-normal->x) );
					*YcT = (float)( pos->y + R*(-normal->y) );

				 }
			

				// Using Snell's law, calculate theta1 (angle between normal and reflected) and theta2 (angle between normal and transmitted)
			no_perturbation:
				theta1 = dot_productT(&dcos_temp,normal);		// cosine of angle between incident in opposite direction and normal (in radians)


				if ( (theta1 > 1.0f) || (theta1 < 0.0f) )	// if incidence angle > 1.57 radian or < 0 radian, then recalculate normal
				{
					// if particle was transmitted, then new normal has to be sampled again
					if(flag_call_transmit == 0)
					{
					mynewnormal:
						normal->x = dcos_temp.x;		// invert dcos of incident vector
						normal->y = dcos_temp.y;
						normal->z = dcos_temp.z;

						RoughSurfaceT(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-90 degrees of inverted dcos.

						mag = sqrt(normal->x*normal->x + normal->y*normal->y);

						// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
						normal->z = 0.0f;			// normal_z of a cylinder is always zero
						normal->x = normal->x/mag;		// re-normalize
						normal->y = normal->y/mag;

						// perturb the normal according to Beta
						if(beta > 0.0f)
							RoughSurfaceT(normal, seed, beta);

						// find the angle between Normal and -Dcos
						cos_newangle = dot_productT(&dcos_temp, normal);
						newangle = acosf(cos_newangle);

						if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 90 degrees from inverted dcos
						{						// keep looping until get a theta within 90 degrees
							if(newnormalctr < 100)
							{
								newnormalctr++;
								goto mynewnormal;			
							}
							else // kill it
							{
								atomicAdd(&num_theta1T,1);
								flag_abs = 1;
								newnormalctr = 0;
								goto baexit;
							}
						}

					}
		
					if(theta1ctr < 100)	// resample max 100 times	
					{
						theta1ctr++;
						goto prpt;
					}
					else	// kill it
					{
						atomicAdd(&num_theta1T,1);
						flag_abs = 1;
						theta1ctr = 0;
						goto baexit;
					}
				}
				else
					theta1 = acosf(theta1);
		

				// check for conditions where photon can only reflect
				if (flag_call_transmit == 1)	// only valid when photon within the column and can transmit outside the column. asin(n1/n2) -> nan
				{
					if (theta1 > asin(n2/n1))	// critical angle condition for TIR
					{
						Pr = 1.0f;		// TIR occurs
						Pt = 0.0f;
					}
			       		else if ( theta1 < epsilon ) 	// theta1 ~= 0, then always reflect
					{
				        	theta1 = 0.00042;       // make theta1 a very smal number, to avoid getting nan probabilities
					        theta2 = asinf((float)(n1/n2)*sin(theta1));     // refracted/transmitted angle in radians

					        // Using Fresnel's law, compute probability of reflection and transmission 
					        Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
				        	Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}
					else    // the ray will transmit
					{
				        	theta2 = asinf((float)(n1/n2)*sin(theta1));     // refracted/transmitted angle in radians
					        
					        // Using Fresnel's law, compute probability of reflection and transmission 
				        	Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
					        Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}

				}
				else if (flag_call_transmit == 0)
				{		
					if((n1/n2) < 1.57f)	// TIR can occur
					{
						if (theta1 > asin(n1/n2))	// critical angle condition for TIR
						{
							Pr = 1.0f;		// TIR occurs
							Pt = 0.0f;
						}
					}
					else if ( theta1 < epsilon )	// theta1 ~= 0, then always reflect
					{
						theta1 = 0.00042;	// make theta1 a very smal number, to avoid getting nan probabilities
						theta2 = asinf((float)(n1/n2)*sin(theta1)); 	// refracted/transmitted angle in radians
	
						// Using Fresnel's law, compute probability of reflection and transmission 
						Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
						Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}
					else	// the ray will transmit
					{
						theta2 = asinf((float)(n2/n1)*sin(theta1));

						// Using Fresnel's law, compute probability of reflection and transmission 
						Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
						Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}

				}


				// normalize Pr and Pt
				temp_norm = Pr + Pt;
				Pr = Pr/temp_norm;
				Pt = Pt/temp_norm;


				if(ranecuT(seed) < Pr)				// reflection
				{
					trans_dir_cosT(dcos, normal, theta1, theta2, 0, mytid, info);


					// condition to check that reflected vector is within 90 degrees from original normal
					angle_oldN_R = dot_productT(&old_normal, dcos);
					angle_oldN_R = acosf(angle_oldN_R);


					if (angle_oldN_R > 1.57f) // > 90 degrees, reperturb the normal
					{
						reperturb_ctr++;

						if(reperturb_ctr < 4)	// maximum 3 times reperturb, else calculate using smooth surface normal (old_normal)
							goto reperturb;
						else
						{
							normal->x = old_normal.x;
							normal->y = old_normal.y;
							normal->z = old_normal.z;

							dcos->x = old_dcos.x;
							dcos->y = old_dcos.y;
							dcos->z = old_dcos.z;

							dcos_temp.x = -dcos->x;
							dcos_temp.y = -dcos->y;
							dcos_temp.z = -dcos->z;

							reperturb_ctr = 0;
				
							if(oldN_Rctr < 25)	// resample max 100 times (25 * reperturb 4 times)	
							{
								oldN_Rctr++;
								goto no_perturbation;
							}
							else	// kill it
							{
								atomicAdd(&num_theta1T,1);
								flag_abs = 1;
								oldN_Rctr = 0;
								goto baexit;
							}

						}

					}

					if (flag_call_transmit == 0)		// it is reflecting between columns, so need to calculate distance using transmit()
					{
						flagCCT = 0;	

						trans_flag = transmitT(pos, dcos, normal, seed, xdetector, ydetector, H, top_absfrac, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, pitch, info, mytid, num_detected_primary, d_max, sensorRefl, flagCCT);

						if (trans_flag == 1)		// photon exited
							flag_abs = 1;
						else if (trans_flag == 0)
							goto prpt;				
					}
				}
				else						// transmission
				{
					trans_dir_cosT(dcos, normal, theta1, theta2, 1, mytid, info);

					if (flag_call_transmit == 1)		// photon travels between columns
					{
						flag_call_transmit = 0;
						flagCCT = 0;

						trans_flag = transmitT(pos, dcos, normal, seed, xdetector, ydetector, H, top_absfrac, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, pitch, info, mytid, num_detected_primary, d_max, sensorRefl, flagCCT);

						if (trans_flag == 1)		// particle exited
							flag_abs = 1;
						else if (trans_flag == 0)	// hits a column
							goto prpt;		// check again to see if it gets reflected or transmitted
					}			
				}
		
			} // else 'prpt ends
		
		} // main else ends

	baexit:
	   return flag_abs;

	}	// CUDA boundary analysis function ends
#else

	int boundary_analysisT(float *normal, float *pos, float *dcos, int* seed, float xdetector, float ydetector, float R, float H, float n1, float n2, float top_absfrac, 
	float beta, float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, struct start_info info, float d_max, 		float sensorRefl, int ydim, int *h_num_detected_prim)
	{

		float dcos_temp[3] = {0.0f};
		float old_normal[3] = {0.0f};
		float old_dcos[3] = {0.0f};
		float Pr = 0.0f, Pt = 0.0f;	// Prob. of reflection and transmission
		float theta1 = 0.0f;
		float theta2 = 0.0f;
		float temp_norm = 0.0f;
		float mag = 0.0f;
		float rr_rnd = 0.0f;
		float theta_rnd = 0.0f;
		float newdepthT = 0.0f;
		float angle_oldN_R = 0.0f;
		float cos_newangle = 0.0f;
		float newangle = 0.0f;
		float cct1 = 0.0f;	// columnar cross talk
		int reperturb_ctr = 0;
		int trans_flag = 0.0f;
		int flag_abs = 0;		// flag - particle got absorbed at top surface or exited during the transmission to another boundary
		int flag_call_transmit = 1;	// flag - particle is going to move within a column (flag = 1) [call isotropic()] or between columns (flag = 0) [call transmit()]
		int theta1ctr=0;	// counter for theta1 > 90 degrees (max resampling 100 times)
		int oldN_Rctr=0;	// counter for angle_oldN_R > 90 degrees (max resampling 25 times)
		int newnormalctr=0;
		int flagCCT = 0;	// flag to indicate in transmit() that the photon needs to cross over

		// determine the coordinates of normal
		if ( (fabs(pos[2] - (float)(H/2.0f)) < epsilon) && (dcos[2] > 0.0f) )	// reached top surface and dir. cosine in z-direction is positive
		{
	
			// top surface absorption - using absorption coefficient 'top_absfrac'
			if ( (top_absfrac > 0.0f) && (ranecuT(seed) < top_absfrac) )	// particle gets absorbed
			{
				flag_abs = 1;
				num_abs_topT++;
			}
			else
			{

				// specular reflector		
				normal[0] = 0.0f;
				normal[1] = 0.0f;
				normal[2] = -1.0f;

				// assign new directional cosines
				dcos[2] = -fabs((ranecuT(seed) * 2.0f) - 1.0f);
				rr_rnd = sqrt(1.0f - dcos[2]*dcos[2]);
				theta_rnd = ranecuT(seed)*twopipen;	

				dcos[0]=rr_rnd*cos(theta_rnd);
				dcos[1]=rr_rnd*sin(theta_rnd);
	
				flag_abs = 0;
			}

		}	
		else 	// compute the normal and check if gets reflected or transmitted
		{	

			// Columnar crosstalk
			newdepthT = H*0.2f;	// top 20% depth CCT=1. considering CsI layer only. NO organic polymer coating.
		
			if( (pos[2] <= H/2.0f) && (pos[2] >= (H/2.0f - newdepthT)) )	// top 20% - 100% cct
			{
				cct1 = 1.0f;
			}
			else if( (pos[2] < (H/2.0f - newdepthT)) && (pos[2] >= 0.0f) )  // from 20% depth to 50% - linear 100% to 50% 
			{
				cct1 = (pos[2]/(2.0f*(H/2.0f - newdepthT))) + 0.5;	
			}
			else if( (pos[2] < 0.0f) && (pos[2] >= (-H/2.0f)) ) // bottom 50% to (-H/2 - 4 um polymer) - 50% to 100% CCT
			{
				cct1 = ( (pos[2] - (-H/2.0f))/(2.0f * (-H/2.0f)) ) + 1.0 ;
			}
	
			if(ranecuT(seed) < cct1)		// columnar cross talk occurs
			{

				// photon crosses over to new column with random orientation. dcos do not change.
				flagCCT = 1;

				trans_flag = transmitT(pos, dcos, normal, seed, xdetector, ydetector, H, top_absfrac, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, info, d_max, sensorRefl, ydim, flagCCT, h_num_detected_prim);

				if (trans_flag == 1)		// photon exited
					flag_abs = 1;
				else if (trans_flag == 0)
				{
					// calculate new column's center coordinates
					XcT = (float)( pos[0] + R*(-normal[0]) );
					YcT = (float)( pos[1] + R*(-normal[1]) );

					flag_abs = 0;
				}
			}
			else
			{

			prpt:

				// within the column
				if(flag_call_transmit == 1)			// photon is currrently within a column with center Xc,Yc
				{
					mag = sqrt( (((XcT)-pos[0]) * ((XcT)-pos[0])) + (((YcT)-pos[1]) * ((YcT)-pos[1])) );
					normal[0] = ((XcT)-pos[0])/mag;
					normal[1] = ((YcT)-pos[1])/mag;
					normal[2] = 0.0f;
		
					if(beta > 0.0f)
						RoughSurfaceT(normal, seed, beta);	// new normal for rough surface

					flag_abs = 0;
				}
				// outside the column
				else if (flag_call_transmit == 0)		// photon is currently between columns and has not entered any column yet. New normal is sampled in the transmit(), so do not calculate normal here.
				{
					// center of new column (obtained by inverting the new normal sampled in transmit() and finding center at distance R from current position)
					XcT = (float)( pos[0] + R*(-normal[0]) );
					YcT = (float)( pos[1] + R*(-normal[1]) );

				        flag_abs = 0;
				}

				// -dcos -> inverted the incident vector to get the smaller angle, else would have to do angle = 180-angle
				dcos_temp[0] = -dcos[0];
				dcos_temp[1] = -dcos[1];
				dcos_temp[2] = -dcos[2];

				old_normal[0] = normal[0];
				old_normal[1] = normal[1];
				old_normal[2] = normal[2];
	
				old_dcos[0] = dcos[0];
				old_dcos[1] = dcos[1];
				old_dcos[2] = dcos[2];


			reperturb:
				normal[0] = old_normal[0];
				normal[1] = old_normal[1];
				normal[2] = old_normal[2];

				dcos[0] = old_dcos[0];
				dcos[1] = old_dcos[1];
				dcos[2] = old_dcos[2];

				dcos_temp[0] = -dcos[0];
				dcos_temp[1] = -dcos[1];
				dcos_temp[2] = -dcos[2];

				if( (flag_call_transmit == 1) && (reperturb_ctr != 0) )
				 {
					if(beta > 0.0f)
						RoughSurfaceT(normal, seed, beta);	// new normal for rough surface
				 }
				if( (flag_call_transmit == 0) && (reperturb_ctr != 0) )
				 {
					if(beta > 0.0f)
						RoughSurfaceT(normal, seed, beta);	// the sampled normal for transmitted photon needs to be perturbed

					// center of new column (obtained by inverting the new normal sampled in transmit() and finding center at distance R from current position)
					XcT = (float)( pos[0] + R*(-normal[0]) );
					YcT = (float)( pos[1] + R*(-normal[1]) );

				 }
			

				// Using Snell's law, calculate theta1 (angle between normal and reflected) and theta2 (angle between normal and transmitted)
			no_perturbation:
				theta1 = dot_productT(dcos_temp,normal);		// cosine of angle between incident in opposite direction and normal (in radians)

				if ( (theta1 > 1.0f) || (theta1 < 0.0f) )	// if incidence angle > 1.57 radian or < 0 radian, then recalculate normal
				{
					// if particle was transmitted, then new normal has to be sampled again
					if(flag_call_transmit == 0)
					{
					mynewnormal:
						normal[0] = dcos_temp[0];		// invert dcos of incident vector
						normal[1] = dcos_temp[1];
						normal[2] = dcos_temp[2];

						RoughSurfaceT(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-90 degrees of inverted dcos.

						mag = sqrt(normal[0]*normal[0] + normal[1]*normal[1]);

						// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
						normal[2] = 0.0f;			// normal_z of a cylinder is always zero
						normal[0] = normal[0]/mag;		// re-normalize
						normal[1] = normal[1]/mag;

						// perturb the normal according to Beta
						if(beta > 0.0f)
							RoughSurfaceT(normal, seed, beta);

						// find the angle between Normal and -Dcos
						cos_newangle = dot_productT(dcos_temp, normal);
						newangle = acosf(cos_newangle);

						if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 90 degrees from inverted dcos
						{
							if(newnormalctr < 100)
							{
								newnormalctr++;
								goto mynewnormal;			
							}
							else // kill it
							{
								num_theta1T++;
								flag_abs = 1;
								newnormalctr = 0;
								goto baexit;
							}				// keep looping until get a theta within 90 degrees
						}

					}

					if(theta1ctr < 100)	
					{
						theta1ctr++;
						goto prpt;
					}
					else	// kill it
					{
						num_theta1T++;
						flag_abs = 1;
						theta1ctr = 0;
						goto baexit;
					}
				}
				else
					theta1 = acosf(theta1);
		

				// check for conditions where photon can only reflect
				if (flag_call_transmit == 1)	// only valid when photon within the column and can transmit outside the column. asin(n1/n2) -> nan
				{
					if (theta1 > asin(n2/n1))	// critical angle condition for TIR
					{
						Pr = 1.0f;		// TIR occurs
						Pt = 0.0f;
					}
			       		else if ( theta1 < epsilon ) 	// theta1 ~= 0, then always reflect
					{
				        	theta1 = 0.00042;       // make theta1 a very smal number, to avoid getting nan probabilities
					        theta2 = asinf((float)(n1/n2)*sin(theta1));     // refracted/transmitted angle in radians

					        // Using Fresnel's law, compute probability of reflection and transmission 
					        Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
				        	Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}
					else    // the ray will transmit
					{
				        	theta2 = asinf((float)(n1/n2)*sin(theta1));     // refracted/transmitted angle in radians
					        
					        // Using Fresnel's law, compute probability of reflection and transmission 
				        	Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
					        Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}

				}
				else if (flag_call_transmit == 0)
				{		
					if((n1/n2) < 1.57f)	// TIR can occur
					{
						if (theta1 > asin(n1/n2))	// critical angle condition for TIR
						{
							Pr = 1.0f;		// TIR occurs
							Pt = 0.0f;
						}
					}
					else if ( theta1 < epsilon )	// theta1 ~= 0, then always reflect
					{
						theta1 = 0.00042;	// make theta1 a very smal number, to avoid getting nan probabilities
						theta2 = asinf((float)(n1/n2)*sin(theta1)); 	// refracted/transmitted angle in radians
	
						// Using Fresnel's law, compute probability of reflection and transmission 
						Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
						Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}
					else	// the ray will transmit
					{
						theta2 = asinf((float)(n2/n1)*sin(theta1));

						// Using Fresnel's law, compute probability of reflection and transmission 
						Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
						Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}

				}


				// normalize Pr and Pt
				temp_norm = Pr + Pt;
				Pr = Pr/temp_norm;
				Pt = Pt/temp_norm;


				if(ranecuT(seed) < Pr)				// reflection
				{
					trans_dir_cosT(dcos, normal, theta1, theta2, 0, info);


					// condition to check that reflected vector is within 90 degrees from original normal
					angle_oldN_R = dot_productT(old_normal, dcos);
					angle_oldN_R = acosf(angle_oldN_R);


					if (angle_oldN_R > 1.57f) // > 90 degrees, reperturb the normal
					{
						reperturb_ctr++;

						if(reperturb_ctr < 4)	// maximum 3 times reperturb, else calculate using smooth surface normal (old_normal)
							goto reperturb;
						else
						{
							normal[0] = old_normal[0];
							normal[1] = old_normal[1];
							normal[2] = old_normal[2];

							dcos[0] = old_dcos[0];
							dcos[1] = old_dcos[1];
							dcos[2] = old_dcos[2];

							dcos_temp[0] = -dcos[0];
							dcos_temp[1] = -dcos[1];
							dcos_temp[2] = -dcos[2];

							reperturb_ctr = 0;


							if(oldN_Rctr < 25)	// max resample 25 times (25*4reperturb = 100 times)
							{
								oldN_Rctr++;
								goto no_perturbation;
							}
							else	// kill it
							{
								num_theta1T++;
								flag_abs = 1;
								oldN_Rctr = 0;
								goto baexit;
							}
						}
					}

					if (flag_call_transmit == 0)		// it is reflecting between columns, so need to calculate distance using transmit()
					{
						flagCCT = 0;

						trans_flag = transmitT(pos, dcos, normal, seed, xdetector, ydetector, H, top_absfrac, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, info, d_max, sensorRefl, ydim, flagCCT, h_num_detected_prim);

						if (trans_flag == 1)		// photon exited
							flag_abs = 1;
						else if (trans_flag == 0)
							goto prpt;				
					}
				}
				else						// transmission
				{
					trans_dir_cosT(dcos, normal, theta1, theta2, 1, info);

					if (flag_call_transmit == 1)		// photon exits current column
					{
						flag_call_transmit = 0;
						flagCCT = 0;

						trans_flag = transmitT(pos, dcos, normal, seed, xdetector, ydetector, H, top_absfrac, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, info, d_max, sensorRefl, ydim, flagCCT, h_num_detected_prim);

						if (trans_flag == 1)		// particle exited
							flag_abs = 1;
						else if (trans_flag == 0)	// hits a column
							goto prpt;		// check again to see if it gets reflected or transmitted
					}			
				}

			} //else prpt ends

		}	// main else ends

	baexit:
	   return flag_abs;

	}	// C boundary analysis function ends
#endif


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// Photon gets transmitted, calculate the new position where it hits next column or boundary
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ int transmitT(float3 *pos, float3 *dcos, float3 *normal, int2* seed, float xdetector, float ydetector, float H, float top_absfrac, float beta, float d_min, 
	int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, size_t pitch, struct start_info *info, int mytid, 
	int *num_detected_primary, float d_max, float sensorRefl, int flagCCT)
	{
		float3 temp_pos = {0.0f};
		float d_nextCol = 0.0f;		// distance to next column	
		float d_top = 0.0f;		// distance to top surface
		float d_bottom = 0.0f;		// distance to bottom surface
		int particle_exit = 0;		// flag to indicate if photon enters another column or gets lost/detected/absorbed
		float newangle = 0.0f;
		float cos_newangle = 0.0f;
		float3 temp_dcos = {0.0f};
		float rr_rnd = 0.0f, theta_rnd = 0.0f;
		float tmp_deno = 0.0f;
		int iii = 0, jjj = 0;
		int reflbtm = 0;

		int newnormalctr=0;	// counter for resampling rough normal to new column (max 100 times, else kill it)
		int newnormalctr2=0;

		temp_pos.x = pos->x;
		temp_pos.y = pos->y;
		temp_pos.z = pos->z;

		temp_dcos.x = -dcos->x;
		temp_dcos.y = -dcos->y;
		temp_dcos.z = -dcos->z;


		if(flagCCT == 1)	// CCT occurs
		{
			// no change in dcos. d_nextcol = 0. new column has random orientation.
			newnormal1:
				normal->x = temp_dcos.x;		// invert dcos of incident vector
				normal->y = temp_dcos.y;
				normal->z = temp_dcos.z;

				RoughSurfaceT(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-90 degrees of inverted dcos.

				tmp_deno = sqrt(normal->x*normal->x + normal->y*normal->y);

				// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
				normal->z = 0.0f;			// normal_z of a cylinder is always zero
				normal->x = normal->x/tmp_deno;		// re-normalize
				normal->y = normal->y/tmp_deno;

				// perturb the normal according to Beta
				if(beta > 0.0f)
					RoughSurfaceT(normal, seed, beta);

				// find the angle between Normal and -Dcos
				cos_newangle = dot_productT(&temp_dcos, normal);
				newangle = acosf(cos_newangle);

				if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 90 degrees from inverted dcos
				 {						// keep looping until get a theta within 90 degrees
					if(newnormalctr < 100)
					{
						newnormalctr++;
						goto newnormal1;			
					}
					else // kill it
					{
						atomicAdd(&num_theta1T,1);
						particle_exit = 1;
						newnormalctr = 0;
						goto exitnow;
					}
						
				 }

				particle_exit = 0;
		}
		else
		{

			// sample distance uniformly between d_min and d_max to next column
			d_nextCol = ranecuT(seed) * (d_max - d_min) + d_min;

			// compute the new position of the photon. 
			pos->x = temp_pos.x + dcos->x * d_nextCol;
			pos->y = temp_pos.y + dcos->y * d_nextCol;
			pos->z = temp_pos.z + dcos->z * d_nextCol;

			// calculate distance to top and bottom surface: if d_top < d_nextCol then photon should reflect from the top surface; else if d_bottom < d_nextCol, photon should get detected.
			d_top = ((H/2.0f) - temp_pos.z)/dcos->z;
			d_bottom  = ((-H/2.0f) - temp_pos.z)/dcos->z;

			// condition to check that pos is within detector boundaries - if true, particle LOST
			if ( (pos->x < epsilon) || (pos->x > xdetector) || (pos->y < epsilon) || (pos->y > ydetector) )
			{
				atomicAdd(&num_lostT, 1);
				particle_exit = 1;
				goto exitnow;
			}

			if ( (pos->z < -H/2.0f) || (pos->z > H/2.0f)  )
				{
					if( (d_top < d_nextCol) && (d_top > epsilon) )
					{
						pos->x = temp_pos.x + dcos->x * d_top;
						pos->y = temp_pos.y + dcos->y * d_top;
						pos->z = H/2.0f;
				
						atomicAdd(&photon_distanceT, d_top);
						particle_exit = 0;			
					}
					else if( (d_bottom < d_nextCol) && (d_bottom > epsilon) )
					{
						pos->x = temp_pos.x + dcos->x * d_bottom;
						pos->y = temp_pos.y + dcos->y * d_bottom;
						pos->z = -H/2.0f;

						atomicAdd(&photon_distanceT, d_bottom);

						// non-ideal sensor - reflects back sensorRefl% of photons into the current column; absorbs rest
						if(ranecuT(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
						{
		
							particle_exit = 0;

							// normal pointing (0,0,1)
							normal->x = 0.0f; normal->y = 0.0f; normal->z = 1.0f;

							// obtain reflected dcos from the bottom (specular reflection; 
							// bottom surface is smooth, so no need to perturb the normal)
							// this condition is called only when photon hits the bottom surface OUTSIDE any column
							trans_dir_cosT(dcos, normal, 0.0f, 0.0f, 0, mytid, info);	// reflection only so refl_theta,trans_theta = 0

							// sample new distance and place new column
							reflbtm = refl_bottomT(pos, dcos, normal, xdetector, ydetector, seed, beta, d_min, H, d_max);

							if(reflbtm == 1)
							{
								particle_exit = 1;
								goto exitnow;
							}

							// if it hits top surface after reflecting back
							if ( (fabs(pos->z - (H/2.0f)) < epsilon) && (dcos->z > 0.0f) )	
							{
								goto mytopsurface;
							}

					

						}
						else
						{
							particle_exit = 1;	
							atomicAdd(&num_detectT, 1);


							iii = floor((pos->x-lbound_x)/pixelsize);	// determine pixel number in x and y direction
							jjj = floor((pos->y-lbound_y)/pixelsize);

							// if the photon gets detected within lower and upper bounds: accumulate the signal contribution
							if( (pos->x <= ubound_x) && (pos->y <= ubound_y) && (pos->x >= lbound_x) && (pos->y >= lbound_y) )
							 {	
								unsigned long long int* current_img = (unsigned long long int*)((char*)myimage + iii * pitch);
								atomicAdd(&current_img[jjj],1);
							 }
		
							atomicAdd(&num_detected_primary[info[mytid].str_histnum-1],1);// start array from 0.str_histnum starts from 1
							
							goto exitnow;
						}	
					}
					else
					{
						atomicAdd(&num_lostT, 1);
						particle_exit = 1;
						goto exitnow;
					}
				}
			else
				atomicAdd(&photon_distanceT, d_nextCol);		// add distance travelled to global variable


			// sample new normal to determine orientation of new column.
		newnormal:
			normal->x = temp_dcos.x;		// invert dcos of incident vector
			normal->y = temp_dcos.y;
			normal->z = temp_dcos.z;

			RoughSurfaceT(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-90 degrees of inverted dcos.

			tmp_deno = sqrt(normal->x*normal->x + normal->y*normal->y);

			// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
			normal->z = 0.0f;			// normal_z of a cylinder is always zero
			normal->x = normal->x/tmp_deno;		// re-normalize
			normal->y = normal->y/tmp_deno;

			// perturb the normal according to Beta
			if(beta > 0.0f)
				RoughSurfaceT(normal, seed, beta);

			// find the angle between Normal and -Dcos
			cos_newangle = dot_productT(&temp_dcos, normal);
			newangle = acosf(cos_newangle);

			if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 90 degrees from inverted dcos
			 {						// keep looping until get a theta within 90 degrees
				if(newnormalctr < 100)
				{
					newnormalctr++;
					goto newnormal;			
				}
				else // kill it
				{
					atomicAdd(&num_theta1T,1);
					particle_exit = 1;
					newnormalctr = 0;
					goto exitnow;
				}
						
			 }
	
			// check if the photon enters another column or got lost (hit detector side)/ reflected (detector top)/ detected (detector bottom)
	
			// hit side of detector?
			if ( (fabs(pos->x-0.0f) < epsilon) || (fabs(pos->x-xdetector) < epsilon) || (fabs(pos->y-0.0f) < epsilon) || (fabs(pos->y-ydetector) < epsilon) )
			{
				atomicAdd(&num_lostT, 1);
				particle_exit = 1;
				goto exitnow;
			}
			

			// hit top?
		     mytopsurface:

			if ( (fabs(pos->z - (H/2.0f)) < epsilon) && (dcos->z > 0.0f) )	// gets specularly reflected or absorbed
			{
				normal->x = 0.0f;
				normal->y = 0.0f;
				normal->z = -1.0f;

				// top surface absorption - using absorption coefficient 'top_absfrac'
				if ( (top_absfrac > 0.0f) && (ranecuT(seed) < top_absfrac) )	// particle gets absorbed
				{
					atomicAdd(&num_abs_topT, 1);
					particle_exit = 1;
					goto exitnow;
				}
				else
				{

					// assign new directional cosines
					dcos->z = -fabs((ranecuT(seed) * 2.0f) - 1.0f);
					rr_rnd = sqrt(1.0f - dcos->z*dcos->z);
					theta_rnd = ranecuT(seed)*twopipen;	
	
					dcos->x=rr_rnd*cos(theta_rnd);
					dcos->y=rr_rnd*sin(theta_rnd);

					temp_pos.x = pos->x;
					temp_pos.y = pos->y;
					temp_pos.z = pos->z;

					temp_dcos.x = -dcos->x;
					temp_dcos.y = -dcos->y;
					temp_dcos.z = -dcos->z;

					// sample distance uniformly between d_min and d_max to next column
					d_nextCol = ranecuT(seed) * (d_max - d_min) + d_min;

					// calculate distance to bottom surface: if d_bottom < d_nextCol, photon should get detected.
					d_bottom  = ((-H/2.0f) - temp_pos.z)/dcos->z;

					// compute the new position of the photon. 
					pos->x = temp_pos.x + dcos->x * d_nextCol;
					pos->y = temp_pos.y + dcos->y * d_nextCol;
					pos->z = temp_pos.z + dcos->z * d_nextCol;

					// condition to check that pos is within detector boundaries - if true, particle LOST
					if ( (pos->x < epsilon) || (pos->x > xdetector) || (pos->y < epsilon) || (pos->y > ydetector) )
					{
						atomicAdd(&num_lostT, 1);
						particle_exit = 1;
						goto exitnow;
					}

					if ( (pos->z < -H/2.0f) || (pos->z > H/2.0f)  )
						{
							if( (d_bottom < d_nextCol) && (d_bottom > epsilon) )
							{
								pos->x = temp_pos.x + dcos->x * d_bottom;
								pos->y = temp_pos.y + dcos->y * d_bottom;
								pos->z = -H/2.0f;

								atomicAdd(&photon_distanceT, d_bottom);

								// non-ideal sensor - reflects back sensorRefl% of photons into the current column; absorbs rest
								if(ranecuT(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
								{
									particle_exit = 0;

									// normal pointing (0,0,1)
									normal->x = 0.0f; normal->y = 0.0f; normal->z = 1.0f;

									// obtain reflected dcos from the bottom (specular reflection; 
									// bottom surface is smooth, so no need to perturb the normal)
									// this condition is called only when photon hits the bottom surface OUTSIDE any column
									trans_dir_cosT(dcos, normal, 0.0f, 0.0f, 0, mytid, info);	// reflection only so refl_theta,trans_theta = 0

									// sample new distance and place new column
									reflbtm = refl_bottomT(pos, dcos, normal, xdetector, ydetector, seed, beta, d_min, H, d_max);

									if(reflbtm == 1)
									{
										particle_exit = 1;
										goto exitnow;
									}

									// if it hits top surface after reflecting back
									if ( (fabs(pos->z - (H/2.0f)) < epsilon) && (dcos->z > 0.0f) )	
									{
										goto mytopsurface;
									}

								}
								else
								{
										particle_exit = 1;
										atomicAdd(&num_detectT, 1);

										iii = floor((pos->x-lbound_x)/pixelsize);// determine pixel number in x and y direction
										jjj = floor((pos->y-lbound_y)/pixelsize);

										// if photon gets detected within lower and upper bounds: accumulate signal contribution
										if( (pos->x <= ubound_x) && (pos->y <= ubound_y) && (pos->x >= lbound_x) && (pos->y >= lbound_y) )
										 {	
											unsigned long long int* current_img = (unsigned long long int*)((char*)myimage + iii * pitch);
											atomicAdd(&current_img[jjj],1);
										 }

										 atomicAdd(&num_detected_primary[info[mytid].str_histnum-1],1);

										goto exitnow;	
								}	
							}
							else
							{
								atomicAdd(&num_lostT, 1);
								particle_exit = 1;
								goto exitnow;
							}
						}
					else
						atomicAdd(&photon_distanceT, d_nextCol);		// add distance travelled to global variable

					// sample new normal to determine orientation of new column.
			  newnormal_TOP:
					normal->x = temp_dcos.x;		// invert dcos of incident vector
					normal->y = temp_dcos.y;
					normal->z = temp_dcos.z;

					RoughSurfaceT(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-90 degrees of inverted dcos.

					tmp_deno = sqrt(normal->x*normal->x + normal->y*normal->y);

					// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
					normal->z = 0.0f;			// normal_z of a cylinder is always zero
					normal->x = normal->x/tmp_deno;		// re-normalize
					normal->y = normal->y/tmp_deno;

					// perturb the normal according to Beta
					if(beta > 0.0f)
						RoughSurfaceT(normal, seed, beta);

					// find the angle between Normal and -Dcos
					cos_newangle = dot_productT(&temp_dcos, normal);
					newangle = acosf(cos_newangle);

					if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 90 degrees from inverted dcos
					 {
						if(newnormalctr2 < 100)	// resample max 100 times
						{
							newnormalctr2++;
							goto newnormal_TOP;			// keep looping until get a theta within 90 degrees		
						}
						else // kill it
						{
							atomicAdd(&num_theta1T,1);
							particle_exit = 1;
							newnormalctr2 = 0;
							goto exitnow;
						}
				
					 }
	
					particle_exit = 0;
				}
			}	// hit top ends
	

			// hit bottom? z of detector can be in the range (-H/2, H/2).
			if ( fabs(pos->z - (-H/2.0f)) < epsilon )	// gets detected
			{
				// non-ideal sensor - reflects back sensorRefl% of photons into the current column; absorbs rest
				if(ranecuT(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
				{
					particle_exit = 0;

					// normal pointing (0,0,1)
					normal->x = 0.0f; normal->y = 0.0f; normal->z = 1.0f;

					// obtain reflected dcos from the bottom (specular reflection; 
					// bottom surface is smooth, so no need to perturb the normal)
					// this condition is called only when photon hits the bottom surface OUTSIDE any column
					trans_dir_cosT(dcos, normal, 0.0f, 0.0f, 0, mytid, info);	// reflection only so refl_theta,trans_theta = 0

					// sample new distance and place new column
					reflbtm = refl_bottomT(pos, dcos, normal, xdetector, ydetector, seed, beta, d_min, H, d_max);

					if(reflbtm == 1)
					{
						particle_exit = 1;
						goto exitnow;
					}

					// if it hits top surface after reflecting back
					if ( (fabs(pos->z - (H/2.0f)) < epsilon) && (dcos->z > 0.0f) )	
					{
						goto mytopsurface;
					}

				}
				else
				{
					atomicAdd(&num_detectT, 1);
					particle_exit = 1;

					iii = floor((pos->x-lbound_x)/pixelsize);	// determine pixel number in x and y direction
					jjj = floor((pos->y-lbound_y)/pixelsize);

					// if the photon gets detected within lower and upper bounds: accumulate the signal contribution
					if( (pos->x <= ubound_x) && (pos->y <= ubound_y) && (pos->x >= lbound_x) && (pos->y >= lbound_y) )
					 {	
						unsigned long long int* current_img = (unsigned long long int*)((char*)myimage + iii * pitch);
						atomicAdd(&current_img[jjj],1);
					 }

					atomicAdd(&num_detected_primary[info[mytid].str_histnum-1],1);	// start array from 0.str_histnum starts from 1
					
					goto exitnow;
				}
			}

		} // else CCT ends
	
	exitnow:
	 return particle_exit;	
	}	// CUDA transmit function ends
#else
	int transmitT(float *pos, float *dcos, float *normal, int* seed, float xdetector, float ydetector, float H, float top_absfrac, float beta, float d_min, int pixelsize, 
	float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, struct start_info info, float d_max, float sensorRefl, int ydim, int flagCCT,
	int *h_num_detected_prim)
	{

		float temp_pos[3] = {0.0f};
		float d_nextCol = 0.0f;		// distance to next column	
		float d_top = 0.0f;		// distance to top surface
		float d_bottom = 0.0f;		// distance to bottom surface
		int particle_exit = 0;		// flag to indicate if photon enters another column or gets lost/detected/absorbed
		float newangle = 0.0f;
		float cos_newangle = 0.0f;
		float temp_dcos[3] = {0.0f};
		float rr_rnd = 0.0f, theta_rnd = 0.0f;
		float tmp_deno = 0.0f;
		int iii = 0, jjj = 0;
		int reflbtm = 0;

		int newnormalctr = 0;
		int newnormalctr2 = 0;

		temp_pos[0] = pos[0];
		temp_pos[1] = pos[1];
		temp_pos[2] = pos[2];

		temp_dcos[0] = -dcos[0];
		temp_dcos[1] = -dcos[1];
		temp_dcos[2] = -dcos[2];

		if(flagCCT == 1)	// CCT occurs
		{
			// no change in dcos. d_nextcol = 0. new column has random orientation.
			newnormal1:
				normal[0] = temp_dcos[0];		// invert dcos of incident vector
				normal[1] = temp_dcos[1];
				normal[2] = temp_dcos[2];

				RoughSurfaceT(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-90 degrees of inverted dcos.

				tmp_deno = sqrt(normal[0]*normal[0] + normal[1]*normal[1]);

				// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
				normal[2] = 0.0f;			// normal_z of a cylinder is always zero
				normal[0] = normal[0]/tmp_deno;		// re-normalize
				normal[1] = normal[1]/tmp_deno;

				// perturb the normal according to Beta
				if(beta > 0.0f)
					RoughSurfaceT(normal, seed, beta);

				// find the angle between Normal and -Dcos
				cos_newangle = dot_productT(temp_dcos, normal);
				newangle = acosf(cos_newangle);

				if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 90 degrees from inverted dcos
				 {						// keep looping until get a theta within 90 degrees
					if(newnormalctr < 100)
					{
						newnormalctr++;
						goto newnormal1;			
					}
					else // kill it
					{
						num_theta1T++;
						particle_exit = 1;
						newnormalctr = 0;
						goto exitnow;
					}
						
				 }

				particle_exit = 0;
		}
		else
		{

			// sample distance uniformly between d_min and d_max to next column
			d_nextCol = ranecuT(seed) * (d_max - d_min) + d_min;

			// compute the new position of the photon. 
			pos[0] = temp_pos[0] + dcos[0] * d_nextCol;
			pos[1] = temp_pos[1] + dcos[1] * d_nextCol;
			pos[2] = temp_pos[2] + dcos[2] * d_nextCol;

			// calculate distance to top and bottom surface: if d_top < d_nextCol then photon should reflect from the top surface; else if d_bottom < d_nextCol, photon should get detected.
			d_top = ((H/2.0f) - temp_pos[2])/dcos[2];
			d_bottom  = ((-H/2.0f) - temp_pos[2])/dcos[2];

			// condition to check that pos is within detector boundaries - if true, particle LOST
			if ( (pos[0] < epsilon) || (pos[0] > xdetector) || (pos[1] < epsilon) || (pos[1] > ydetector) )
			{
				num_lostT++;
				particle_exit = 1;
				goto exitnow;
			}

			if ( (pos[2] < -H/2.0f) || (pos[2] > H/2.0f)  )
				{
					if( (d_top < d_nextCol) && (d_top > epsilon) )
					{
						pos[0] = temp_pos[0] + dcos[0] * d_top;
						pos[1] = temp_pos[1] + dcos[1] * d_top;
						pos[2] = H/2.0f;
				
						photon_distanceT = photon_distanceT + d_top;
						particle_exit = 0;			
					}
					else if( (d_bottom < d_nextCol) && (d_bottom > epsilon) )
					{
						pos[0] = temp_pos[0] + dcos[0] * d_bottom;
						pos[1] = temp_pos[1] + dcos[1] * d_bottom;
						pos[2] = -H/2.0f;

						photon_distanceT = photon_distanceT + d_bottom;

						// non-ideal sensor - reflects back sensorRefl% of photons into the current column; absorbs rest
						if(ranecuT(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
						{
		
							particle_exit = 0;

							// normal pointing (0,0,1)
							normal[0] = 0.0f; normal[1] = 0.0f; normal[2] = 1.0f;

							// obtain reflected dcos from the bottom (specular reflection; 
							// bottom surface is smooth, so no need to perturb the normal)
							// this condition is called only when photon hits the bottom surface OUTSIDE any column
							trans_dir_cosT(dcos, normal, 0.0f, 0.0f, 0, info);	// reflection only so refl_theta,trans_theta = 0

							// sample new distance and place new column
							reflbtm = refl_bottomT(pos, dcos, normal, xdetector, ydetector, seed, beta, d_min, H, d_max);

							if(reflbtm == 1)
							{
								particle_exit = 1;
								goto exitnow;
							}


							// if it hits top surface after reflecting back
							if ( (fabs(pos[2] - (H/2.0f)) < epsilon) && (dcos[2] > 0.0f) )	
							{
								goto mytopsurface;
							}

						}
						else
						{
							particle_exit = 1;	
							num_detectT++;


							iii = floor((pos[0]-lbound_x)/pixelsize);	// determine pixel number in x and y direction
							jjj = floor((pos[1]-lbound_y)/pixelsize);

							// if the photon gets detected within lower and upper bounds: accumulate the signal contribution
							if( (pos[0] <= ubound_x) && (pos[1] <= ubound_y) && (pos[0] >= lbound_x) && (pos[1] >= lbound_y) )
							 {	
								outputimage_.tempimageopt[iii][jjj]++;
							 }

							goto exitnow;
						}	
					}
					else
					{
						num_lostT++;
						particle_exit = 1;
						goto exitnow;
					}
				}
			else
				photon_distanceT = photon_distanceT + d_nextCol;		// add distance travelled to global variable


			// sample new normal to determine orientation of new column.
		newnormal:
			normal[0] = temp_dcos[0];		// invert dcos of incident vector
			normal[1] = temp_dcos[1];
			normal[2] = temp_dcos[2];

			RoughSurfaceT(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-90 degrees of inverted dcos.

			tmp_deno = sqrt(normal[0]*normal[0] + normal[1]*normal[1]);

			// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
			normal[2] = 0.0f;			// normal_z of a cylinder is always zero
			normal[0] = normal[0]/tmp_deno;		// re-normalize
			normal[1] = normal[1]/tmp_deno;

			// perturb the normal according to Beta
			if(beta > 0.0f)
				RoughSurfaceT(normal, seed, beta);

			// find the angle between Normal and -Dcos
			cos_newangle = dot_productT(temp_dcos, normal);
			newangle = acosf(cos_newangle);

			if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 90 degrees from inverted dcos
			 {						// keep looping until get a theta within 90 degrees
				if(newnormalctr < 100)
				{
					newnormalctr++;
					goto newnormal;			
				}
				else // kill it
				{
					num_theta1T++;
					particle_exit = 1;
					newnormalctr = 0;
					goto exitnow;
				}
						
			 }
	
			// check if the photon enters another column or got lost (hit detector side)/ reflected (detector top)/ detected (detector bottom)
	
			// hit side of detector?
			if ( (fabs(pos[0]-0.0f) < epsilon) || (fabs(pos[0]-xdetector) < epsilon) || (fabs(pos[1]-0.0f) < epsilon) || (fabs(pos[1]-ydetector) < epsilon) )
			{
				num_lostT++;
				particle_exit = 1;
				goto exitnow;
			}
		

			// hit top?
		     mytopsurface:

			if ( (fabs(pos[2] - (H/2.0f)) < epsilon) && (dcos[2] > 0.0f) )	// gets specularly reflected or absorbed
			{
				normal[0] = 0.0f;
				normal[1] = 0.0f;
				normal[2] = -1.0f;

				// top surface absorption - using absorption coefficient 'top_absfrac'
				if ( (top_absfrac > 0.0f) && (ranecuT(seed) < top_absfrac) )	// particle gets absorbed
				{
					num_abs_topT++;
					particle_exit = 1;
					goto exitnow;
				}
				else
				{

					// assign new directional cosines
					dcos[2] = -fabs((ranecuT(seed) * 2.0f) - 1.0f);
					rr_rnd = sqrt(1.0f - dcos[2]*dcos[2]);
					theta_rnd = ranecuT(seed)*twopipen;	
	
					dcos[0]=rr_rnd*cos(theta_rnd);
					dcos[1]=rr_rnd*sin(theta_rnd);

					temp_pos[0] = pos[0];
					temp_pos[1] = pos[1];
					temp_pos[2] = pos[2];

					temp_dcos[0] = -dcos[0];
					temp_dcos[1] = -dcos[1];
					temp_dcos[2] = -dcos[2];

					// sample distance uniformly between d_min and d_max to next column
					d_nextCol = ranecuT(seed) * (d_max - d_min) + d_min;

					// calculate distance to bottom surface: if d_bottom < d_nextCol, photon should get detected.
					d_bottom  = ((-H/2.0f) - temp_pos[2])/dcos[2];

					// compute the new position of the photon. 
					pos[0] = temp_pos[0] + dcos[0] * d_nextCol;
					pos[1] = temp_pos[1] + dcos[1] * d_nextCol;
					pos[2] = temp_pos[2] + dcos[2] * d_nextCol;

					// condition to check that pos is within detector boundaries - if true, particle LOST
					if ( (pos[0] < epsilon) || (pos[0] > xdetector) || (pos[1] < epsilon) || (pos[1] > ydetector) )
					{
						num_lostT++;
						particle_exit = 1;
						goto exitnow;
					}

					if ( (pos[2] < -H/2.0f) || (pos[2] > H/2.0f)  )
						{
							if( (d_bottom < d_nextCol) && (d_bottom > epsilon) )
							{
								pos[0] = temp_pos[0] + dcos[0] * d_bottom;
								pos[1] = temp_pos[1] + dcos[1] * d_bottom;
								pos[2] = -H/2.0f;

								photon_distanceT = photon_distanceT + d_bottom;

								// non-ideal sensor - reflects back sensorRefl% of photons into the current column; absorbs rest
								if(ranecuT(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
								{

									particle_exit = 0;		
			
									// normal pointing (0,0,1)
									normal[0] = 0.0f; normal[1] = 0.0f; normal[2] = 1.0f;

									// obtain reflected dcos from the bottom (specular reflection; 
									// bottom surface is smooth, so no need to perturb the normal)
									// this condition is called only when photon hits the bottom surface OUTSIDE any column
									trans_dir_cosT(dcos, normal, 0.0f, 0.0f, 0, info);	// reflection only so refl_theta,trans_theta = 0

									// sample new distance and place new column
									reflbtm = refl_bottomT(pos, dcos, normal, xdetector, ydetector, seed, beta, d_min, H, d_max);

									if(reflbtm == 1)
									{
										particle_exit = 1;
										goto exitnow;
									}

									// if it hits top surface after reflecting back
									if ( (fabs(pos[2] - (H/2.0f)) < epsilon) && (dcos[2] > 0.0f) )	
									{
										goto mytopsurface;
									}

								}
								else
								{
										particle_exit = 1;
										num_detectT++;

										iii = floor((pos[0]-lbound_x)/pixelsize);
										jjj = floor((pos[1]-lbound_y)/pixelsize);

										// if photon gets detected within lower and upper bounds: accumulate signal contribution
										if( (pos[0] <= ubound_x) && (pos[1] <= ubound_y) && (pos[0] >= lbound_x) && (pos[1] >= lbound_y) )
										 {	
											outputimage_.tempimageopt[iii][jjj]++;
										 }

										goto exitnow;	
								}	
							}
							else
							{
								num_lostT++;
								particle_exit = 1;
								goto exitnow;
							}
						}
					else
						photon_distanceT = photon_distanceT + d_nextCol;		// add distance travelled to global variable

					// sample new normal to determine orientation of new column.
			  newnormal_TOP:
					normal[0] = temp_dcos[0];		// invert dcos of incident vector
					normal[1] = temp_dcos[1];
					normal[2] = temp_dcos[2];

					RoughSurfaceT(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-90 degrees of inverted dcos.

					tmp_deno = sqrt(normal[0]*normal[0] + normal[1]*normal[1]);

					// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
					normal[2] = 0.0f;			// normal_z of a cylinder is always zero
					normal[0] = normal[0]/tmp_deno;		// re-normalize
					normal[1] = normal[1]/tmp_deno;

					// perturb the normal according to Beta
					if(beta > 0.0f)
						RoughSurfaceT(normal, seed, beta);

					// find the angle between Normal and -Dcos
					cos_newangle = dot_productT(temp_dcos, normal);
					newangle = acosf(cos_newangle);

					if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 90 degrees from inverted dcos
					 {
						if(newnormalctr2 < 100)	// resample max 100 times
						{
							newnormalctr2++;
							goto newnormal_TOP;			// keep looping until get a theta within 90 degrees		
						}
						else // kill it
						{
							num_theta1T++;
							particle_exit = 1;
							newnormalctr2 = 0;
							goto exitnow;
						}
				
					 }
	
					particle_exit = 0;
				}
			}	// hit top ends
	

			// hit bottom? z of detector can be in the range (-H/2, H/2).
			if ( fabs(pos[2] - (-H/2.0f)) < epsilon )	// gets detected
			{
				// non-ideal sensor - reflects back sensorRefl% of photons into the current column; absorbs rest
				if(ranecuT(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
				{
					particle_exit = 0;		
	
					// normal pointing (0,0,1)
					normal[0] = 0.0f; normal[1] = 0.0f; normal[2] = 1.0f;

					// obtain reflected dcos from the bottom (specular reflection; 
					// bottom surface is smooth, so no need to perturb the normal)
					// this condition is called only when photon hits the bottom surface OUTSIDE any column
					trans_dir_cosT(dcos, normal, 0.0f, 0.0f, 0, info);	// reflection only so refl_theta,trans_theta = 0

					// sample new distance and place new column
					reflbtm = refl_bottomT(pos, dcos, normal, xdetector, ydetector, seed, beta, d_min, H, d_max);

					if(reflbtm == 1)
					{
						particle_exit = 1;
						goto exitnow;
					}

					// if it hits top surface after reflecting back
					if ( (fabs(pos[2] - (H/2.0f)) < epsilon) && (dcos[2] > 0.0f) )	
					{
						goto mytopsurface;
					}

				}
				else
				{
					num_detectT++;
					particle_exit = 1;

					iii = floor((pos[0]-lbound_x)/pixelsize);	// determine pixel number in x and y direction
					jjj = floor((pos[1]-lbound_y)/pixelsize);

					// if the photon gets detected within lower and upper bounds: accumulate the signal contribution
					if( (pos[0] <= ubound_x) && (pos[1] <= ubound_y) && (pos[0] >= lbound_x) && (pos[1] >= lbound_y) )
					 {	
						outputimage_.tempimageopt[iii][jjj]++;
					 }

					goto exitnow;
				}
			}

		} // else CCT ends

	exitnow:
	 return particle_exit;	
	}	// C transmit function ends
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// photon reflects from sensor_plane or bottom surface, when in between columns. 
// Obtains the next column where it hits.
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ int refl_bottomT(float3 *pos, float3 *dcos, float3 *normal, float xdetector, float ydetector, int2* seed, float beta, float d_min, float H, float d_max)
	{

		float3 temp_pos, temp_dcos;
		float d_nextCol=0.0f, d_top=0.0f;
		float tmp_deno=0.0f, cos_newangle=0.0f, newangle=0.0f;
		int pexit=0;

		int newnormalctr=0;

		temp_pos.x = pos->x;
		temp_pos.y = pos->y;
		temp_pos.z = pos->z;

		temp_dcos.x = -dcos->x;
		temp_dcos.y = -dcos->y;
		temp_dcos.z = -dcos->z;

		// sample distance uniformly between d_min and d_max to next column
		d_nextCol = ranecuT(seed) * (d_max - d_min) + d_min;

		// calculate distance to bottom surface: if d_bottom < d_nextCol, photon should get detected.
		d_top  = ((H/2.0f) - temp_pos.z)/dcos->z;

		// compute the new position of the photon. 
		pos->x = temp_pos.x + dcos->x * d_nextCol;
		pos->y = temp_pos.y + dcos->y * d_nextCol;
		pos->z = temp_pos.z + dcos->z * d_nextCol;

		// condition to check that pos is within detector boundaries - if true, particle LOST
		if ( (pos->x < epsilon) || (pos->x > xdetector) || (pos->y < epsilon) || (pos->y > ydetector) )
		{
			atomicAdd(&num_lostT, 1);
			pexit = 1;
			goto myexit;
		}

		if ( (pos->z > H/2.0f)  )		// check if photon's new z position is above top surface
		{
				if( (d_top < d_nextCol) && (d_top > epsilon) )		// photon will hit top surface
				{
					pos->x = temp_pos.x + dcos->x * d_top;
					pos->y = temp_pos.y + dcos->y * d_top;
					pos->z = H/2.0f;
				
					atomicAdd(&photon_distanceT, d_top);
					pexit = 0;			
				}
				else
				{
					atomicAdd(&num_lostT, 1);
					pexit = 1;
					goto myexit;
				}
		}
		else
		{
			atomicAdd(&photon_distanceT, d_nextCol);		// add distance travelled to global variable

			// sample new normal to determine orientation of new column.
		  	mynewnormal:
				normal->x = temp_dcos.x;		// invert dcos of incident vector
				normal->y = temp_dcos.y;
				normal->z = temp_dcos.z;

				RoughSurfaceT(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-90 degrees of inverted dcos.

				tmp_deno = sqrt(normal->x*normal->x + normal->y*normal->y);

				// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
				normal->z = 0.0f;			// normal_z of a cylinder is always zero
				normal->x = normal->x/tmp_deno;		// re-normalize
				normal->y = normal->y/tmp_deno;

				// perturb the normal according to Beta
				RoughSurfaceT(normal, seed, beta);

				// find the angle between Normal and -Dcos
				cos_newangle = dot_productT(&temp_dcos, normal);
				newangle = acosf(cos_newangle);

				if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 90 degrees from inverted dcos
				 {

					if(newnormalctr < 100)	// resample max 100 times
					{
						newnormalctr++;
						goto mynewnormal;			// keep looping until get a theta within 90 degrees	
					}
					else // kill it
					{
						atomicAdd(&num_theta1T,1);
						pexit = 1;
						newnormalctr = 0;
						goto myexit;
					}
				
				 }
	
				pexit = 0;
		}

	myexit:

	 return pexit;
	}	// CUDA refl_bottom function ends
#else
	int refl_bottomT(float *pos, float *dcos, float *normal, float xdetector, float ydetector, int* seed, float beta, float d_min, float H, float d_max)
	{
		float temp_pos[3], temp_dcos[3];
		float d_nextCol=0.0f, d_top=0.0f;
		float tmp_deno=0.0f, cos_newangle=0.0f, newangle=0.0f;
		int pexit=0;
	
		int newnormalctr = 0;

		temp_pos[0] = pos[0];
		temp_pos[1] = pos[1];
		temp_pos[2] = pos[2];

		temp_dcos[0] = -dcos[0];
		temp_dcos[1] = -dcos[1];
		temp_dcos[2] = -dcos[2];

		// sample distance uniformly between d_min and d_max to next column
		d_nextCol = ranecuT(seed) * (d_max - d_min) + d_min;

		// calculate distance to bottom surface: if d_bottom < d_nextCol, photon should get detected.
		d_top  = ((H/2.0f) - temp_pos[2])/dcos[2];

		// compute the new position of the photon. 
		pos[0] = temp_pos[0] + dcos[0] * d_nextCol;
		pos[1] = temp_pos[1] + dcos[1] * d_nextCol;
		pos[2] = temp_pos[2] + dcos[2] * d_nextCol;

		// condition to check that pos is within detector boundaries - if true, particle LOST
		if ( (pos[0] < epsilon) || (pos[0] > xdetector) || (pos[1] < epsilon) || (pos[1] > ydetector) )
		{
			num_lostT++;
			pexit = 1;
			goto myexit;
		}

		if ( (pos[2] > H/2.0f)  )		// check if photon's new z position is above top surface
		{
				if( (d_top < d_nextCol) && (d_top > epsilon) )		// photon will hit top surface
				{
					pos[0] = temp_pos[0] + dcos[0] * d_top;
					pos[1] = temp_pos[1] + dcos[1] * d_top;
					pos[2] = H/2.0f;
				
					photon_distanceT = photon_distanceT + d_top;
					pexit = 0;			
				}
				else
				{
					num_lostT++;
					pexit = 1;
					goto myexit;
				}
		}
		else
		{
			photon_distanceT = photon_distanceT + d_nextCol;		// add distance travelled to global variable

			// sample new normal to determine orientation of new column.
		  	mynewnormal:

				normal[0] = temp_dcos[0];		// invert dcos of incident vector
				normal[1] = temp_dcos[1];
				normal[2] = temp_dcos[2];

				RoughSurfaceT(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-90 degrees of inverted dcos.

				tmp_deno = sqrt(normal[0]*normal[0] + normal[1]*normal[1]);

				// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
				normal[2] = 0.0f;			// normal_z of a cylinder is always zero
				normal[0] = normal[0]/tmp_deno;		// re-normalize
				normal[1] = normal[1]/tmp_deno;

				// perturb the normal according to Beta
				RoughSurfaceT(normal, seed, beta);

				// find the angle between Normal and -Dcos
				cos_newangle = dot_productT(temp_dcos, normal);
				newangle = acosf(cos_newangle);

				if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 90 degrees from inverted dcos
				 {

					if(newnormalctr < 100)	// resample max 100 times
					{
						newnormalctr++;
						goto mynewnormal;			// keep looping until get a theta within 90 degrees	
					}
					else // kill it
					{
						num_theta1T++;
						pexit = 1;
						newnormalctr = 0;
						goto myexit;
					}
				
				 }
	
				pexit = 0;
		}

	myexit:

	 return pexit;
	}	// C refl_bottom function ends
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// calculate dot product of two vectors
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ inline float dot_productT(float3 *aa, float3 *b)
	{
		float result = 0.0f;

		result = aa->x*b->x + aa->y*b->y + aa->z*b->z;

	  return result;
	}
#else
	float dot_productT(float *aa, float *b)
	{
		float result = 0.0f;

		result = aa[0]*b[0] + aa[1]*b[1] + aa[2]*b[2];

	  return result;
	}
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// compute directional cosines of transmitted vector
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ inline void trans_dir_cosT(float3 *dcos, float3 *normal, float refl_theta, float trans_theta, int flag_ref, int mytid, struct start_info *info)
	{
		float cos_angle = 0.0f;
		float norm = 0.0f;
		float3 dcos_temp = {0.0f};

		dcos_temp.x = -dcos->x;
		dcos_temp.y = -dcos->y;
		dcos_temp.z = -dcos->z;
	
		cos_angle = dot_productT(&dcos_temp,normal);	// cosine of angle between incident in opposite direction and normal

		if (flag_ref == 0)				// reflection
		{
				dcos->x = 2.0f*cos_angle*normal->x + dcos->x;  // specular ray
				dcos->y = 2.0f*cos_angle*normal->y + dcos->y;
				dcos->z = 2.0f*cos_angle*normal->z + dcos->z;
		}
		else if (flag_ref == 1)				// transmission	
		{
			 dcos->x= -normal->x*cos(trans_theta)-(sin(trans_theta)/sin(refl_theta))*(dcos->x+(cos_angle*normal->x));
			 dcos->y= -normal->y*cos(trans_theta)-(sin(trans_theta)/sin(refl_theta))*(dcos->y+(cos_angle*normal->y));
			 dcos->z= -normal->z*cos(trans_theta)-(sin(trans_theta)/sin(refl_theta))*(dcos->z+(cos_angle*normal->z));
		}

		norm = sqrt(dcos->x*dcos->x + dcos->y*dcos->y + dcos->z*dcos->z);

		if ((norm < (1.0f - epsilon)) || (norm > (1.0f + epsilon)))
		 {
			dcos->x = dcos->x/norm;
			dcos->y = dcos->y/norm;
			dcos->z = dcos->z/norm;
		 } 

	return;	
	}
#else
	void trans_dir_cosT(float *dcos, float *normal, float refl_theta, float trans_theta, int flag_ref, struct start_info info)
	{
		float cos_angle = 0.0f;
		float norm = 0.0f;
		float dcos_temp[3] = {0.0f};

		dcos_temp[0] = -dcos[0];
		dcos_temp[1] = -dcos[1];
		dcos_temp[2] = -dcos[2];
	
		cos_angle = dot_productT(dcos_temp,normal);	// cosine of angle between incident in opposite direction and normal

		if (flag_ref == 0)				// reflection
		{
				dcos[0] = 2.0f*cos_angle*normal[0] + dcos[0];  // specular ray
				dcos[1] = 2.0f*cos_angle*normal[1] + dcos[1];
				dcos[2] = 2.0f*cos_angle*normal[2] + dcos[2];
		}
		else if (flag_ref == 1)				// transmission	
		{
			 dcos[0]= -normal[0]*cos(trans_theta)-(sin(trans_theta)/sin(refl_theta))*(dcos[0]+(cos_angle*normal[0]));
			 dcos[1]= -normal[1]*cos(trans_theta)-(sin(trans_theta)/sin(refl_theta))*(dcos[1]+(cos_angle*normal[1]));
			 dcos[2]= -normal[2]*cos(trans_theta)-(sin(trans_theta)/sin(refl_theta))*(dcos[2]+(cos_angle*normal[2]));
		}

		norm = sqrt(dcos[0]*dcos[0] + dcos[1]*dcos[1] + dcos[2]*dcos[2]);

		if ((norm < (1.0f - epsilon)) || (norm > (1.0f + epsilon)))
		 {
			dcos[0] = dcos[0]/norm;
			dcos[1] = dcos[1]/norm;
			dcos[2] = dcos[2]/norm;
		 } 

	return;	
	}
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// add roughness to the surface of the column according to roughness coefficient 'beta'
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ inline void RoughSurfaceT(float3 *normal, int2* seed, float beta)
	{

		float theta = 0.0f;
		float status = 0.0f;
		float rr = 0.0f;
		float3 normalpert = {0.0f};
		float3 rough_normal = {0.0f};
		float normalize_base = 0.0f;

		// generate the perturbation vector
		status = ranecuT(seed);
		normalpert.z = 2.0f*status - 1.0f;
		rr = sqrt(1.0f - status*status);
		status = ranecuT(seed);
		theta = status * 2.0f * pi;

		normalpert.x = rr * cos(theta);
		normalpert.y = rr * sin(theta);

		// normalize the perturbed vector
		normalize_base = sqrt( pow(normalpert.x,2) + pow(normalpert.y,2) + pow(normalpert.z,2) );
	
		normalpert.x = normalpert.x/normalize_base;
		normalpert.y = normalpert.y/normalize_base;
		normalpert.z = normalpert.z/normalize_base;


		// rough normal = beta*perturbed + original normal
		rough_normal.x = beta * normalpert.x + normal->x;	
		rough_normal.y = beta * normalpert.y + normal->y;
		rough_normal.z = beta * normalpert.z + normal->z;

		// normalize new normal
		normalize_base = sqrt( pow(rough_normal.x,2) + pow(rough_normal.y,2) + pow(rough_normal.z,2) );

		normal->x = rough_normal.x/normalize_base; 
		normal->y = rough_normal.y/normalize_base;
		normal->z = rough_normal.z/normalize_base;


	return;
	}	// CUDA RoughSurface function ends
#else
	void RoughSurfaceT(float *normal, int* seed, float beta)
	{

		float theta = 0.0f;
		float status = 0.0f;
		float rr = 0.0f;
		float normalpert[3] = {0.0f};
		float rough_normal[3] = {0.0f};
		float normalize_base = 0.0f;

		// generate the perturbation vector
		status = ranecuT(seed);
		normalpert[2] = 2.0f*status - 1.0f;
		rr = sqrt(1.0f - status*status);
		status = ranecuT(seed);
		theta = status * 2.0f * pi;

		normalpert[0] = rr * cos(theta);
		normalpert[1] = rr * sin(theta);

		// normalize the perturbed vector
		normalize_base = sqrt( pow(normalpert[0],2) + pow(normalpert[1],2) + pow(normalpert[2],2) );
	
		normalpert[0] = normalpert[0]/normalize_base;
		normalpert[1] = normalpert[1]/normalize_base;
		normalpert[2] = normalpert[2]/normalize_base;


		// rough normal = beta*perturbed + original normal
		rough_normal[0] = beta * normalpert[0] + normal[0];	
		rough_normal[1] = beta * normalpert[1] + normal[1];
		rough_normal[2] = beta * normalpert[2] + normal[2];

		// normalize new normal
		normalize_base = sqrt( pow(rough_normal[0],2) + pow(rough_normal[1],2) + pow(rough_normal[2],2) );

		normal[0] = rough_normal[0]/normalize_base; 
		normal[1] = rough_normal[1]/normalize_base;
		normal[2] = rough_normal[2]/normalize_base;


	return;
	}	// C RoughSurface function ends
#endif


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// determine if the photon gets detected
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ inline int detectionT(float3 *pos, float H, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, 
	size_t pitch, struct start_info *info, int mytid, int *num_detected_primary, float sensorRefl, float d_min, int2* seed, float3 *dcos, float3 *normal, float bulk_abscoeff, 
	float R, float xdetector, float ydetector, unsigned long long int mynum_rebound, float *XcT, float *YcT)
	{
		int result = 0;
		int ii = 0, jj = 0;
		int absflag = 0;

		// equation of plane is z = -H/2
		// if a point satisfies above equation, it is detected

		if (fabs(pos->z - (float)(-H/2.0f)) < epsilon) 
		 {
			// non-ideal sensor - reflects back sensorRefl% of photons into the current column; absorbs rest
			if(ranecuT(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
			{

				// normal pointing (0,0,1)
				normal->x = 0.0f; 
				normal->y = 0.0f; 
				normal->z = 1.0f;

				// obtain reflected dcos from the bottom (specular reflection; bottom surface is smooth, no need to perturb the normal)
				// this function is called only when photon hits bottom of a column
				trans_dir_cosT(dcos, normal, 0.0f, 0.0f, 0, mytid, info); // as their is only reflection so refl_theta,trans_theta = 0

				// using above calculated dcos, move the particle within the column
				absflag = isotropicT(pos, dcos, seed, bulk_abscoeff, R, H, xdetector, ydetector, &info[mytid], mynum_rebound, XcT, YcT, mytid);

				if(absflag == 1)	// if gets absorbed in bulk, photon exits
					result = 1;
				else
					result = 0;

			}
			else	// photon absorbed or detected  
			{
				result = 1;
				atomicAdd(&num_detectT, 1);
		
				ii = floor((pos->x-lbound_x)/pixelsize);	// determine pixel number in x and y direction
				jj = floor((pos->y-lbound_y)/pixelsize);

				// if the photon gets detected within lower and upper bounds: accumulate the signal contribution
				if( (pos->x <= ubound_x) && (pos->y <= ubound_y) && (pos->x >= lbound_x) && (pos->y >= lbound_y) )
				 {	
					unsigned long long int* current_img = (unsigned long long int*)((char*)myimage + ii * pitch);
					atomicAdd(&current_img[jj],1);
				 }

				atomicAdd(&num_detected_primary[info[mytid].str_histnum-1],1);	// start array from 0.str_histnum starts from 1
			}
			 
		 }
		else
		    	result = 0;
	  
	//exit1:
	 return result;
	}	// CUDA detection function ends
#else
	int detectionT(float *pos, float H, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, struct start_info info, 		float sensorRefl, float d_min, int* seed, float *dcos, float *normal, float bulk_abscoeff, float R, float xdetector, float ydetector, unsigned long long int mynum_rebound, 
	int ydim, int *h_num_detected_prim)
	{

		int result = 0;
		int ii = 0, jj = 0;
		int absflag = 0;


		// equation of plane is z = -H/2
		// if a point satisfies above equation, it is detected

		if (fabs(pos[2] - (float)(-H/2.0f)) < epsilon) 
		 {
			// non-ideal sensor - reflects back sensorRefl% of photons into the current column; absorbs rest
			if(ranecuT(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
			{

				// normal pointing (0,0,1)
				normal[0] = 0.0f; 
				normal[1] = 0.0f; 
				normal[2] = 1.0f;

				// obtain reflected dcos from the bottom (specular reflection; bottom surface is smooth, so no need to perturb the normal)
				// this function is called only when photon hits bottom of a column
				trans_dir_cosT(dcos, normal, 0.0f, 0.0f, 0, info); // as their is only reflection so made refl_theta, trans_theta = 0

				// using above calculated dcos, move the particle within the column
				absflag = isotropicT(pos, dcos, seed, bulk_abscoeff, R, H, xdetector, ydetector, info, mynum_rebound);

				if(absflag == 1)	// if gets absorbed in bulk, photon exits
					result = 1;
				else
					result = 0;

			}
			else	// photon absorbed or detected  
			{
				result = 1;
				num_detectT++;
		
				ii = floor((pos[0]-lbound_x)/pixelsize);	// determine pixel number in x and y direction
				jj = floor((pos[1]-lbound_y)/pixelsize);

				// if the photon gets detected within lower and upper bounds: accumulate the signal contribution
				if( (pos[0] <= ubound_x) && (pos[1] <= ubound_y) && (pos[0] >= lbound_x) && (pos[1] >= lbound_y) )
				 {	
					outputimage_.tempimageopt[ii][jj]++;
				 }

			}
			 
		 }
		else
		    	result = 0;
	  
	 return result;
	}
#endif


////////////////////////////////////////////////////////////////////////////////
//! Initialize the pseudo-random number generator (PRNG) RANECU to a position
//! far away from the previous history (leap frog technique).
//!
//! Each calculated seed initiates a consecutive and disjoint sequence of
//! pseudo-random numbers with length LEAP_DISTANCE, that can be used to
//! in a parallel simulation (Sequence Splitting parallelization method).
//! The basic equation behind the algorithm is:
//!    S(i+j) = (a**j * S(i)) MOD m = [(a**j MOD m)*S(i)] MOD m  ,
//! which is described in:
//!   P L'Ecuyer, Commun. ACM 31 (1988) p.742
//!
//! This function has been adapted from "seedsMLCG.f", see:
//!   A Badal and J Sempau, Computer Physics Communications 175 (2006) p. 440-450
//!
//!       @param[in] history   Particle bach number.
//!       @param[in] seed_input   Initial PRNG seed input (used to initiate both MLCGs in RANECU).
//!       @param[out] seed   Initial PRNG seeds for the present history.
//!
////////////////////////////////////////////////////////////////////////////////
// -- Upper limit of the number of random values sampled in a single track:
#define  LEAP_DISTANCE    1000
// -- Multipliers and moduli for the two MLCG in RANECU:
#define  a1_RANECU       40014
#define  m1_RANECU  2147483563
#define  a2_RANECU       40692
#define  m2_RANECU  2147483399

#ifdef USING_CUDA
	__device__ inline void init_PRNGT(int history_batch, int histories_per_thread, int seed_input, int2* seed)
	{
	  // -- Move the RANECU generator to a unique position for the current batch of histories:
	  //    I have to use an "unsigned long long int" value to represent all the simulated histories in all previous batches
	  //    The maximum unsigned long long int value is ~1.8e19: if history >1.8e16 and LEAP_DISTANCE==1000, 'leap' will overflow.
	  // **** 1st MLCG:
	  unsigned long long int leap = ((unsigned long long int)(history_batch+1))*(histories_per_thread*LEAP_DISTANCE);
	  int y = 1;
	  int z = a1_RANECU;
	  // -- Calculate the modulo power '(a^leap)MOD(m)' using a divide-and-conquer algorithm adapted to modulo arithmetic
	  for(;;)
	  {
	    // (A2) Halve n, and store the integer part and the residue
	    if (0!=(leap&01))  // (bit-wise operation for MOD(leap,2), or leap%2 ==> proceed if leap is an odd number)  Equivalent: t=(short)(leap%2);
	    {
	      leap >>= 1;     // Halve n moving the bits 1 position right. Equivalent to:  leap=(leap/2);  
	      y = abMODmT(m1_RANECU,z,y);      // (A3) Multiply y by z:  y = [z*y] MOD m
	      if (0==leap) break;         // (A4) leap==0? ==> finish
	    }
	    else           // (leap is even)
	    {
	      leap>>= 1;     // Halve leap moving the bits 1 position right. Equivalent to:  leap=(leap/2);
	    }
	    z = abMODmT(m1_RANECU,z,z);        // (A5) Square z:  z = [z*z] MOD m
	  }
	  // AjMODm1 = y;                 // Exponentiation finished:  AjMODm = expMOD = y = a^j

	  // -- Compute and display the seeds S(i+j), from the present seed S(i), using the previously calculated value of (a^j)MOD(m):
	  //         S(i+j) = [(a**j MOD m)*S(i)] MOD m
	  //         S_i = abMODmT(m,S_i,AjMODm)
	  seed->x = abMODmT(m1_RANECU, seed_input, y);     // Using the input seed as the starting seed

	  // **** 2nd MLCG (repeating the previous calculation for the 2nd MLCG parameters):
	  leap = ((unsigned long long int)(history_batch+1))*(histories_per_thread*LEAP_DISTANCE);
	  y = 1;
	  z = a2_RANECU;
	  for(;;)
	  {
	    // (A2) Halve n, and store the integer part and the residue
	    if (0!=(leap&01))  // (bit-wise operation for MOD(leap,2), or leap%2 ==> proceed if leap is an odd number)  Equivalent: t=(short)(leap%2);
	    {
	      leap >>= 1;     // Halve n moving the bits 1 position right. Equivalent to:  leap=(leap/2);
	      y = abMODmT(m2_RANECU,z,y);      // (A3) Multiply y by z:  y = [z*y] MOD m
	      if (0==leap) break;         // (A4) leap==0? ==> finish
	    }
	    else           // (leap is even)
	    {
	      leap>>= 1;     // Halve leap moving the bits 1 position right. Equivalent to:  leap=(leap/2);
	    }
	    z = abMODmT(m2_RANECU,z,z);        // (A5) Square z:  z = [z*z] MOD m
	  }
	  // AjMODm2 = y;
	  seed->y = abMODmT(m2_RANECU, seed_input, y);     // Using the input seed as the starting seed
	}
#else
	void init_PRNGT(int history_batch, int histories_per_thread, int seed_input, int* seed)
	{
	  // -- Move the RANECU generator to a unique position for the current batch of histories:
	  //    I have to use an "unsigned long long int" value to represent all the simulated histories in all previous batches
	  //    The maximum unsigned long long int value is ~1.8e19: if history >1.8e16 and LEAP_DISTANCE==1000, 'leap' will overflow.
	  // **** 1st MLCG:
	  unsigned long long int leap = ((unsigned long long int)(history_batch+1))*(histories_per_thread*LEAP_DISTANCE);
	  int y = 1;
	  int z = a1_RANECU;
	  // -- Calculate the modulo power '(a^leap)MOD(m)' using a divide-and-conquer algorithm adapted to modulo arithmetic
	  for(;;)
	  {

	    // (A2) Halve n, and store the integer part and the residue
	    if (0!=(leap&01))  // (bit-wise operation for MOD(leap,2), or leap%2 ==> proceed if leap is an odd number)  !!DeBuG!! OLD: t=(short)(leap%2);
	    {
	      leap >>= 1;     // Halve n moving the bits 1 position right. Equivalent to:  leap=(leap/2); 
	      y = abMODmT(m1_RANECU,z,y);      // (A3) Multiply y by z:  y = [z*y] MOD m
	      if (0==leap) break;         // (A4) leap==0? ==> finish
	    }
	    else           // (leap is even)
	    {
	      leap>>= 1;     // Halve leap moving the bits 1 position right. Equivalent to:  leap=(leap/2);
	    }
	    z = abMODmT(m1_RANECU,z,z);        // (A5) Square z:  z = [z*z] MOD m
	  }
	  // AjMODm1 = y;                 // Exponentiation finished:  AjMODm = expMOD = y = a^j

	  // -- Compute and display the seeds S(i+j), from the present seed S(i), using the previously calculated value of (a^j)MOD(m):
	  //         S(i+j) = [(a**j MOD m)*S(i)] MOD m
	  //         S_i = abMODmT(m,S_i,AjMODm)
	  seed[0] = abMODmT(m1_RANECU, seed_input, y);     // Using the input seed as the starting seed

	  // **** 2nd MLCG (repeating the previous calculation for the 2nd MLCG parameters):
	  leap = ((unsigned long long int)(history_batch+1))*(histories_per_thread*LEAP_DISTANCE);
	  y = 1;
	  z = a2_RANECU;
	  for(;;)
	  {
	    // (A2) Halve n, and store the integer part and the residue
	    if (0!=(leap&01))  // (bit-wise operation for MOD(leap,2), or leap%2 ==> proceed if leap is an odd number)  !!DeBuG!! OLD: t=(short)(leap%2);
	    {
	      leap >>= 1;     // Halve n moving the bits 1 position right. Equivalent to:  leap=(leap/2); 
	      y = abMODmT(m2_RANECU,z,y);      // (A3) Multiply y by z:  y = [z*y] MOD m
	      if (0==leap) break;         // (A4) leap==0? ==> finish
	    }
	    else           // (leap is even)
	    {
	      leap>>= 1;     // Halve leap moving the bits 1 position right. Equivalent to:  leap=(leap/2);
	    }
	    z = abMODmT(m2_RANECU,z,z);        // (A5) Square z:  z = [z*z] MOD m
	  }
	  // AjMODm2 = y;
	  seed[1] = abMODmT(m2_RANECU, seed_input, y);     // Using the input seed as the starting seed

	}
#endif


/////////////////////////////////////////////////////////////////////
//!  Calculate "(a1*a2) MOD m" with 32-bit integers and avoiding   **
//!  the possible overflow, using the Russian Peasant approach     **
//!  modulo m and the approximate factoring method, as described   **
//!  in:  L'Ecuyer and Cote, ACM Trans. Math. Soft. 17 (1991)      **
//!                                                                **
//!  This function has been adapted from "seedsMLCG.f", see:       **
//!  Badal and Sempau, Computer Physics Communications 175 (2006)  **
//!                                                                **
//!    Input:          0 < a1 < m                                  **
//!                    0 < a2 < m                                  **
//!                                                                **
//!    Return value:  (a1*a2) MOD m                                **
//!                                                                **
/////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ inline int abMODmT(int m, int a, int s)
	{
	  // CAUTION: the input parameters are modified in the function but should not be returned to the calling function! (pass by value!)
	  int q, k;
	  int p = -m;            // p is always negative to avoid overflow when adding

	  // ** Apply the Russian peasant method until "a =< 32768":
	  while (a>32768)        // We assume '32' bit integers (4 bytes): 2^(('32'-2)/2) = 32768
	  {
	    if (0!=(a&1))        // Store 's' when 'a' is odd     Equivalent code:   if (1==(a%2))
	    {
	      p += s;
	      if (p>0) p -= m;
	    }
	    a >>= 1;             // Half a (move bits 1 position right)   Equivalent code: a = a/2;
	    s = (s-m) + s;       // Double s (MOD m)
	    if (s<0) s += m;     // (s is always positive)
	  }

	  // ** Employ the approximate factoring method (a is small enough to avoid overflow):
	  q = (int) m / a;
	  k = (int) s / q;
	  s = a*(s-k*q)-k*(m-q*a);
	  while (s<0)
	    s += m;

	  // ** Compute the final result:
	  p += s;
	  if (p<0) p += m;

	  return p;
	}
#else
	int abMODmT(int m_par, int a_par, int s_par)
	{
	  // CAUTION: the input parameters are modified in the function but should not be returned to the calling function! (pass by value!)   !!DeBuG!!
	  int mval,aval,sval;
	  mval=m_par; aval=a_par; sval=s_par;
	  
	  int qval, kval;
	  int pval = -mval;            // p is always negative to avoid overflow when adding

	  // ** Apply the Russian peasant method until "a =< 32768":
	  while (aval>32768)        // We assume '32' bit integers (4 bytes): 2^(('32'-2)/2) = 32768
	  {
	    if (0!=(aval&1))        // Store 's' when 'a' is odd    !!DeBuG!! OLD code:   if (1==(a%2))
	    {
	      pval += sval;
	      if (pval>0) pval -= mval;
	    }
	    aval >>= 1;             // Half a (move bits 1 position right)        
	    sval = (sval-mval) + sval;       // float s (MOD m)
	    if (sval<0) sval += mval;     // (s is always positive)
	  }

	  // ** Employ the approximate factoring method (a is small enough to avoid overflow):
	  qval = (int) mval / aval;
	  kval = (int) sval / qval;
	  sval = aval*(sval-kval*qval)-kval*(mval-qval*aval);
	  while (sval<0)
	    sval += mval;

	  // ** Compute the final result:
	  pval += sval;
	  if (pval<0) pval += mval;

	  return pval;
	}
#endif


////////////////////////////////////////////////////////////////////////////////
//! Pseudo-random number generator (PRNG) RANECU returning a float value
//! (single precision version).
//!
//!       @param[in,out] seed   PRNG seed (seed kept in the calling function and updated here).
//!       @return   PRN double value in the open interval (0,1)
//!
////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ inline float ranecuT(int2* seed)
	{
	//return (float(seed->x%100)*0.01f+0.005f)  ;

	  int i1 = (int)(seed->x/53668);
	  seed->x = 40014*(seed->x-i1*53668)-i1*12211;

	  int i2 = (int)(seed->y/52774);
	  seed->y = 40692*(seed->y-i2*52774)-i2*3791;

	  if (seed->x < 0) seed->x += 2147483563;
	  if (seed->y < 0) seed->y += 2147483399;

	  i2 = seed->x-seed->y;
	  if (i2 < 1) i2 += 2147483562;

	  return (__int2float_rn(i2)*4.65661305739e-10f);        // 4.65661305739e-10 == 1/2147483563

	}
#else
	float ranecuT(int* seed)
	{
	  int i1 = (int)(seed[0]/53668);
	  seed[0] = 40014*(seed[0]-i1*53668)-i1*12211;

	  int i2 = (int)(seed[1]/52774);
	  seed[1] = 40692*(seed[1]-i2*52774)-i2*3791;

	  if (seed[0] < 0) seed[0] += 2147483563;
	  if (seed[1] < 0) seed[1] += 2147483399;

	  i2 = seed[0]-seed[1];
	  if (i2 < 1) i2 += 2147483562;

	  const float USCALE = 1.0/2147483563.0;       
	  return ((float)(i2*USCALE));

	}
#endif

